#include "hip/hip_runtime.h"
#include <iomanip>
#include <iostream>
#include <cstdlib>
#include <math.h>
#include <hip/hip_runtime.h>
#include <hipDNN.h>
#include <stdio.h>
#include <string.h>
#include <opencv2/opencv.hpp>

using namespace cv;
//***********************************************
//**���Ŀ� �� �������� GPU �޸� SizeȮ�� �ʿ�**
//***********************************************

//***********************
//**�ɼ� ���� ��� �߰�**
//***********************

//********************
//**alpha, beta ����**
//********************
using namespace std;

int main()
{
	//**********
	//**Handle**
	//**********
	hipdnnHandle_t cudnn;
	hipdnnCreate(&cudnn);

	//********
	//**����**
	//********

	//�Էº���
	const int ImageNum = 1;
	const int FeatureNum = 3;
	const int FeatureHeight = 32;
	const int FeatureWidth = 32;

	//********
	//**�Է�**
	//********
	Mat img = imread("D:\\DataSet\\cifar\\test\\0_cat.png");	// �̹��������� �о� �鿩 Mat �������� �����Ű��
	unsigned char* imgd = img.data;
	//�Է���� ����
	float Input[ImageNum][FeatureNum][FeatureHeight][FeatureWidth];

	//�Է���� ����
	for (int i = 0; i < 3; i++)
	{
		for (int y = 0; y < 32; y++)
		{
			for (int x = 0; x < 32; x++)
			{
				Input[0][i][y][x] = imgd[3 * 32  * y + 3 * x + i];
			}
		}
	}

	//GPU�� �Է���� �޸� �Ҵ� �� �� ����
	float * dev_Input;
	hipMalloc((void**)&dev_Input, sizeof(Input));
	hipMemcpy(dev_Input, Input, sizeof(Input), hipMemcpyHostToDevice);

	//�Է���� ����ü ����, �Ҵ�, �ʱ�ȭ
	hipdnnTensorDescriptor_t in_desc; //�Է� ������ �� ������ ���� �ִ� ����ü�� ����Ű�� ���� ������
	hipdnnCreateTensorDescriptor(&in_desc); // 4D tensor ����ü ��ü ����
	hipdnnSetTensor4dDescriptor( // 4D tensor ����ü �ʱ�ȭ �Լ�
		/*tensorDesc,*/ in_desc, 
		/*format,*/HIPDNN_TENSOR_NCHW, 
		/*dataType,*/HIPDNN_DATA_FLOAT,
		/*Number of images*/ImageNum,
		/*C*/FeatureNum,
		/*H*/FeatureHeight,
		/*W*/FeatureWidth);

	//************************
	//************************
	//**Feedforward �������**
	//************************
	//************************

	//*******************
	//**Convolution����**
	//*******************

	//���� ������ ���� - ���� ���� �����ϵ���
	const int filt_n = 1;
	const int filt_c = 3;
	const int filt_h = 8;
	const int filt_w = 8;


	//���� ����
	float Filter[filt_n][filt_c][filt_h][filt_w];

	//���� ����
	for (int ch = 0; ch < filt_c; ch++)
	{
	for (int row = 0; row < filt_h; row++)
	{
		for (int col = 0; col < filt_w; col++)
		{
			Filter[0][ch][col][row] = (float)((col + row) % 3);
		}

	}
	}
	//GPU�� ������� ����
	float * dev_Filt;
	hipMalloc((void**)&dev_Filt, sizeof(float) * filt_n * filt_c * filt_h * filt_w);
	hipMemcpy(dev_Filt, Filter, sizeof(float) * filt_n * filt_c * filt_h * filt_w, hipMemcpyHostToDevice);

	//���ͱ���ü ����, ����, �ʱ�ȭ
	hipdnnFilterDescriptor_t filt_desc; // ���� ������ ���� ����ü�� ����Ű�� ���� ������
	hipdnnCreateFilterDescriptor(&filt_desc); // ���� ����ü ���� 
	hipdnnSetFilter4dDescriptor( // 4d filter ����ü ��ü �ʱ�ȭ
		/*filterDesc,*/filt_desc, 
		/*dataType,*/HIPDNN_DATA_FLOAT, 
		/*format,*/HIPDNN_TENSOR_NCHW,
		/*Number of output feature maps*/filt_n,
		/*Number of input feature maps.*/filt_c,
		/*Height of each filter.*/filt_h,
		/*Width of each filter.*/filt_w);



	//Convolution ���꿡���� ���� ���� - ���� ���� �����ϵ���
	const int pad_h = 2; //padding ����
	const int pad_w = 2; //padding ����
	const int str_h = 4; //stride ����
	const int str_w = 4; //stride ����
	const int dil_h = 1; //dilated ����
	const int dil_w = 1; //dilated ����

	//Convolution ����ü ���� �� �Ҵ�
	hipdnnConvolutionDescriptor_t conv_desc; // Convolution ������ ���� ������ ���� ����ü ������ 
	hipdnnCreateConvolutionDescriptor(&conv_desc); // Convolution ����ü ��ü ����
	hipdnnSetConvolution2dDescriptor(//
		/*convDesc,*/conv_desc,
		/*zero-padding height*/pad_h,
		/*zero-padding width*/pad_w,
		/*Vertical filter stride*/str_h,
		/*Horizontal filter stride*/str_w,
		/*Filter height dilation*/dil_h,
		/*Filter width dilation*/dil_w,
		/*mode*/HIPDNN_CONVOLUTION,
		/*computeType*/HIPDNN_DATA_FLOAT);


	//Convolution ��� ������� ���� �� �Ҵ�
	hipdnnTensorDescriptor_t out_conv_desc;
	hipdnnCreateTensorDescriptor(&out_conv_desc);

	//Convolution ������ ������ ����
	int out_conv_n;
	int out_conv_c;
	int out_conv_h;
	int out_conv_w;

	hipdnnGetConvolution2dForwardOutputDim( // �־��� ����, tensor, convolution ����ü ������ ��������, 2D convolution ��꿡 ���� 4d tensor�� ��� ���� ������ ��ȯ, �� ��� output�� ���� �� �ε�...  
		/*convolution descriptor*/conv_desc, 
		/*tensor descriptor*/in_desc,
		/*filter descriptor*/filt_desc,
		/*Output. Number of output images*/&out_conv_n,
		/*Output. Number of output feature maps per image.*/&out_conv_c,
		/*Output. Height of each output feature map.*/&out_conv_h,
		/*Output. Width of each output feature map.*/&out_conv_w);

	//outputDim = 1 + ( inputDim + 2*pad - (((filterDim-1)*dilation)+1) )/convolutionStride

	int outputDim = 1 + (FeatureHeight + 2 * pad_h - filt_h) / str_h;

	//Convolution��� ����
	float Output_Conv[ImageNum][FeatureNum][8][8];


	//GPU�� Convolution ��� ��� �Ҵ�
	float * dev_Output_Conv;
	hipMalloc((void**)&dev_Output_Conv, sizeof(float) * out_conv_c * out_conv_h * out_conv_n * out_conv_w);

	//Convolution ����ü �ʱ�ȭ
	hipdnnSetTensor4dDescriptor(out_conv_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT,
		ImageNum, FeatureNum, outputDim, outputDim);

	//�Է°� ����, ������� �е�, ��Ʈ���̵尡 ���� ���� �־������� ���� ���� �˰����� ���������� �˾Ƴ���
	hipdnnConvolutionFwdAlgo_t alg;
	hipdnnGetConvolutionForwardAlgorithm(
		cudnn,
		in_desc,
		filt_desc,
		conv_desc,
		out_conv_desc,
		HIPDNN_CONVOLUTION_FWD_PREFER_FASTEST,
		0,
		&alg);
	cout << "Fastest algorithm for conv0 = " << alg << endl;

	//Conv ���� ������ũ�� �˾Ƴ��� �� �� ���� �޸� �Ҵ� �߰�
	size_t WS_size = 0;
	hipdnnGetConvolutionForwardWorkspaceSize(// This function returns the amount of GPU memory workspace
		cudnn, in_desc, filt_desc, conv_desc, out_conv_desc, alg, &WS_size);

	size_t * dev_WS;
	hipMalloc((void**)&dev_WS, WS_size);

	//����
	float alpha = 1.0;
	float beta = 0.0;

	hipdnnConvolutionForward(
		cudnn, 
		&alpha, 
		in_desc, 
		dev_Input, 
		filt_desc, 
		dev_Filt, 
		conv_desc,
		alg, 
		dev_WS, 
		WS_size, 
		&beta, 
		out_conv_desc, 
		dev_Output_Conv);

	//Convolution��� GPU�� ����
	hipMemcpy(Output_Conv, dev_Output_Conv,
		sizeof(float) * out_conv_n * out_conv_c * out_conv_h * out_conv_w, hipMemcpyDeviceToHost);

	
	//********
	//**Bias**
	//********
	beta = 1.0f;

	//Bias ��� ������� ����
	float Output_Bias[ImageNum][FeatureNum][FeatureHeight][FeatureWidth];

	//bias �� ����
	float biasValue[filt_n] = { 10.0f };

	//GPU�� bias�� ����
	float * dev_Bias;
	hipMalloc((void**)&dev_Bias, sizeof(float));
	hipMemcpy(dev_Bias, biasValue, sizeof(float), hipMemcpyHostToDevice);

	//bias��� ������� ����, �Ҵ�
	hipdnnTensorDescriptor_t bias_desc;
	hipdnnCreateTensorDescriptor(&bias_desc);
	hipdnnSetTensor4dDescriptor(
		bias_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, 1, filt_n, 1, 1);

	//bias ���� ���� 
	hipdnnAddTensor(cudnn, &alpha, bias_desc, dev_Bias,
		&beta, /*input -> output*/out_conv_desc, /*input -> output*/dev_Output_Conv);

	//Bias�� ���
	hipMemcpy(Output_Bias, dev_Output_Conv,
		sizeof(float) * ImageNum * FeatureNum * FeatureHeight * FeatureWidth, hipMemcpyDeviceToHost);

	//***********************
	//**Actiovation Funtion**
	//***********************

	beta = 0.0;

	//Activation Function ����ü ���� �� �Ҵ� 
	hipdnnActivationDescriptor_t act_desc;
	hipdnnCreateActivationDescriptor(&act_desc);

	//Activation Function ���� ���� - ���� ���������ϵ���
	hipdnnActivationMode_t Activation_Function;
	Activation_Function = HIPDNN_ACTIVATION_RELU;
	hipdnnSetActivationDescriptor(act_desc, Activation_Function, HIPDNN_PROPAGATE_NAN, 0);

	//Activation Function �޸� GPU�� ����
	float * dev_Output_Act;
	hipMalloc((void**)&dev_Output_Act, sizeof(float) * 32 * 32*3);


	//Activatin Function �������
	hipdnnActivationForward(
		cudnn, act_desc, &alpha, out_conv_desc, dev_Output_Conv,
		&beta, out_conv_desc, dev_Output_Act);

	//Activation Function ����� ���� ���
	float Output_Activation[ImageNum][FeatureNum][FeatureHeight][FeatureWidth];
	hipMemcpy(Output_Activation, dev_Output_Act,
		sizeof(float) * ImageNum * FeatureNum * FeatureHeight * FeatureWidth,
		hipMemcpyDeviceToHost);

	//***************
	//**Pooling����**
	//***************

	//Pooling ���꿡�� ���� ���� - ���� ���� �����ϵ���
	beta = 0.0;

	const int pool_wind_h = 2;
	const int pool_wind_w = 2;
	const int pool_pad_h = 0;
	const int pool_pad_w = 0;
	const int pool_strd_w = 2;
	const int pool_strd_h = 2;

	//Pooling ����ü ���� �� �Ҵ� - ���� Pooling ��� ���� �����ϵ���
	hipdnnPoolingDescriptor_t pool_desc;
	hipdnnCreatePoolingDescriptor(&pool_desc);
	hipdnnSetPooling2dDescriptor(pool_desc, HIPDNN_POOLING_MAX, HIPDNN_PROPAGATE_NAN,
		pool_wind_h, pool_wind_w, pool_pad_h, pool_pad_w, pool_strd_h, pool_strd_w);

	//Pooling ������� ��� ���� �� �Ҵ�
	hipdnnTensorDescriptor_t out_pool_desc;
	hipdnnCreateTensorDescriptor(&out_pool_desc);

	//Pooling ������ ������
	int out_pool_n;
	int out_pool_c;
	int out_pool_h;
	int out_pool_w;

	//Pooling ������ ������ ����
	hipdnnGetPooling2dForwardOutputDim(pool_desc, out_conv_desc,
		&out_pool_n, &out_pool_c, &out_pool_h, &out_pool_w);

	//GPU�� Pooling ������ �޸��Ҵ�
	float * dev_Output_Pool;
	hipMalloc((void**)&dev_Output_Pool,
		sizeof(float) * out_pool_n * out_pool_c * out_pool_h * out_pool_w);

	//Pooling ������� ����ü �ʱ�ȭ
	hipdnnSetTensor4dDescriptor(out_pool_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT,
		out_pool_n, out_pool_c, out_pool_h, out_pool_w);

	//Pooling���� ����
	hipdnnPoolingForward(cudnn, pool_desc, &alpha, out_conv_desc, dev_Output_Act,
		&beta, out_pool_desc, dev_Output_Pool);

	//Pooling���
	float Output_Pool[ImageNum][FeatureNum][(FeatureHeight + 2 * pool_pad_h) / pool_strd_h][(FeatureWidth + 2 * pool_pad_w) / pool_strd_w];
	hipMemcpy(Output_Pool, dev_Output_Pool,
		sizeof(float) * out_pool_n * out_pool_c * out_pool_h * out_pool_w, hipMemcpyDeviceToHost);


	//*******************
	//**Fully Connected**
	//*******************

	//Weights ����
	float Weights[1][1][16][32];

	//Weights ����
	for (int row = 0; row < 16; row++)
	{
		for (int col = 0; col < 32; col++)
		{
			Weights[0][0][row][col] = (float)((row + col) % 4) * 0.2;
			//Weights[0][0][row][col + 3] = 0.99f;
		}
	}

	//GPU�� Weights��� ����
	float * dev_weights;
	hipMalloc((void**)&dev_weights,
		sizeof(float) * 1 * 1 * 16 * 32);
	hipMemcpy(dev_weights, Weights,
		sizeof(float) * 1 * 1 * 16 * 32, hipMemcpyHostToDevice);

	//Weights�� ���� Filter ����ü ���� �� �Ҵ�
	hipdnnFilterDescriptor_t weights_desc;
	hipdnnCreateFilterDescriptor(&weights_desc);
	hipdnnSetFilter4dDescriptor(
		weights_desc, HIPDNN_DATA_FLOAT, HIPDNN_TENSOR_NCHW, 1, 1, 16, 32);

	//Fully Connected�� ���� Convolution ����ü ���� �� �Ҵ�
	hipdnnConvolutionDescriptor_t fc_desc;
	hipdnnCreateConvolutionDescriptor(&fc_desc);
	hipdnnSetConvolution2dDescriptor(fc_desc, 0, 16, 1, 16, 1, 1, HIPDNN_CONVOLUTION, HIPDNN_DATA_FLOAT);

	//Fully Connected ���� ��� ������� ����ü ����
	hipdnnTensorDescriptor_t out_fc_desc;
	hipdnnCreateTensorDescriptor(&out_fc_desc);

	//Fully Connected ������ ������ ����
	int out_fc_n;
	int out_fc_c;
	int out_fc_h;
	int out_fc_w;

	hipdnnGetConvolution2dForwardOutputDim(
		fc_desc, out_fc_desc, weights_desc, &out_fc_n, &out_fc_c, &out_fc_h, &out_fc_w);

	//FC ������ ����
	float Output_FC[1][1][1][2];

	//GPU�� FC ������ �Ҵ�
	float *dev_Output_FC;
	hipMalloc((void**)&dev_Output_FC, sizeof(float) * 1 * 1 * 1 * 2);

	//FC ����ü �ʱ�ȭ
	hipdnnSetTensor4dDescriptor(out_fc_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, 1, 1, 1, 2);


	//�Է°� ����, ������� �е�, ��Ʈ���̵尡 ���� ���� �־������� ���� ���� �˰����� ���������� �˾Ƴ���
	hipdnnConvolutionFwdAlgo_t alg2;
	hipdnnGetConvolutionForwardAlgorithm(
		cudnn,
		out_pool_desc,
		weights_desc,
		fc_desc,
		out_fc_desc,
		HIPDNN_CONVOLUTION_FWD_PREFER_FASTEST,
		0,
		&alg2);
	cout << "Fastest algorithm for FC = " << alg2 << endl;



	//FC ����ũ�� �Ҵ� �� ����
	size_t WS_size2 = 0;
	hipdnnGetConvolutionForwardWorkspaceSize(
		cudnn, out_pool_desc, weights_desc, fc_desc, out_fc_desc, alg2, &WS_size2);

	size_t * dev_WS2;
	hipMalloc((void**)&dev_WS2, WS_size2);

	//Fully Connected ���� 
	hipdnnConvolutionForward(
		cudnn, &alpha, out_pool_desc, dev_Output_Pool, weights_desc, dev_weights, fc_desc,
		alg2, dev_WS2, WS_size2, &beta, out_fc_desc, dev_Output_FC);

	//FC ����� CPU�� ����
	hipMemcpy(Output_FC, dev_Output_FC, sizeof(float) * 1 * 1 * 1 * 2, hipMemcpyDeviceToHost);


	//*************************
	//**Fully Conncected Bias**
	//*************************
	beta = 1.0f;

	//FC bias ��� ����
	float Output_FC_Bias[1][1][1][2];

	//FC bias��
	float biasValueFC[1] = { -5.0f };

	//GPU�� FC bias�� ����
	float * dev_Bias_FC;
	hipMalloc((void**)&dev_Bias_FC, sizeof(float));
	hipMemcpy(dev_Bias_FC, biasValueFC, sizeof(float), hipMemcpyHostToDevice);


	//FC Softmax ����ü - �� �� ��������?
	hipdnnTensorDescriptor_t out_fc_soft_desc;
	hipdnnCreateTensorDescriptor(&out_fc_soft_desc);
	hipdnnSetTensor4dDescriptor(out_fc_soft_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, 1, 1, 1, 1);


	//bias ���� ����
	hipdnnAddTensor(cudnn, &alpha, out_fc_soft_desc, dev_Bias_FC, &beta, out_fc_desc, dev_Output_FC);
	hipMemcpy(Output_FC_Bias, dev_Output_FC, sizeof(float) * 1 * 2, hipMemcpyDeviceToHost);


	//***********
	//**Softmax**
	//***********
	beta = 0.0;

	float OutSoft[1][1][1][2];
	float * dev_Output_Softmax;
	hipMalloc((void**)&dev_Output_Softmax, sizeof(float) * 1 * 2);


	hipdnnSoftmaxForward(cudnn, HIPDNN_SOFTMAX_FAST, HIPDNN_SOFTMAX_MODE_INSTANCE/*�� �κ� �ſ� �߿� - INSTANCE�� �ؾ� �ٷ� �̺а��*/,
		&alpha, out_fc_desc, dev_Output_FC, &beta, out_fc_desc, dev_Output_Softmax);

	hipMemcpy(OutSoft, dev_Output_Softmax, sizeof(float) * 1 * 2, hipMemcpyDeviceToHost);

	//*********
	//**Error**
	//*********

	float error = -log(OutSoft[0][0][0][0]);


	//****************************
	//****************************
	//**Backpropagation �������**
	//****************************
	//****************************

	//***************************
	//**Softmax Backpropagation** - p - y
	//***************************


	//������� ����
	float SoftBack[1][1][1][2];

	//GPU �޸� �Ҵ�
	float * dif_Soft_Back;
	hipMalloc((void**)&dif_Soft_Back, sizeof(float) * 1 * 2);

	//����ü ���� �� �ʱ�ȭ
	hipdnnTensorDescriptor_t dif_soft_desc;
	hipdnnCreateTensorDescriptor(&dif_soft_desc);
	hipdnnSetTensor4dDescriptor(dif_soft_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, 1, 1, 1, 2);


	//delta

	float DivSoft[2] = { 0.01f, 0.0f };

	float * dev_Output_Soft_Back;
	hipMalloc((void**)&dev_Output_Soft_Back, sizeof(float) * 1 * 2);



	float * dev_dif_Softmax;
	hipMalloc((void**)&dev_dif_Softmax, sizeof(float) * 1 * 2);
	hipMemcpy(dev_dif_Softmax, DivSoft, sizeof(float) * 1 * 2, hipMemcpyHostToDevice);


	hipdnnTensorDescriptor_t dif_soft_back;
	hipdnnCreateTensorDescriptor(&dif_soft_back);
	hipdnnSetTensor4dDescriptor(dif_soft_back, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, 1, 1, 1, 2);



	hipdnnSoftmaxBackward(cudnn, HIPDNN_SOFTMAX_ACCURATE, /*****�̺κ� �ſ� �߿�*****/HIPDNN_SOFTMAX_MODE_INSTANCE, &alpha,
		dif_soft_back, dev_Output_Softmax, dif_soft_back, dev_dif_Softmax, &beta, dif_soft_back, dev_Output_Soft_Back);

	hipMemcpy(SoftBack, dev_Output_Soft_Back, sizeof(float) * 1 * 2, hipMemcpyDeviceToHost);


	//*********************************
	//**Fully Connected Bias Backward**
	//*********************************

	/*

	//�������
	float FCbiasBack[1][1][1][1];

	//GPU �޸�
	float * dev_FC_bias_Back;
	hipMalloc((void**)&dev_FC_bias_Back, sizeof(float));

	hipdnnConvolutionBackwardBias(cudnn, &alpha, out_fc_desc, dev_Output_Softmax, &beta, bias_desc, dev_FC_bias_Back);

	hipMemcpy(FCbiasBack, dev_FC_bias_Back, sizeof(float), hipMemcpyDeviceToHost);

	*/

	//**********************************
	//**Fully Connected Backpropagtion**
	//**********************************

	//������� ����
	float FCBack[1][3][16][32];

	//GPU�� �޸� �Ҵ�
	float * dev_Filter_Gradient;
	hipMalloc((void**)&dev_Filter_Gradient, sizeof(float) * 3 * 16 * 32);

	// Workspace
	size_t WS_size3 = 0;
	hipdnnGetConvolutionBackwardDataWorkspaceSize(cudnn, weights_desc, dif_soft_back, fc_desc, out_pool_desc, HIPDNN_CONVOLUTION_BWD_DATA_ALGO_0, &WS_size3);

	//GPU�� workspace �޸� �Ҵ�
	size_t * dev_WS3;
	hipMalloc((void**)&dev_WS3, WS_size3);

	//Fully Connected Backpropagation delta
	hipdnnConvolutionBackwardFilter(cudnn, &alpha,
		out_pool_desc, dev_Output_Pool, dif_soft_back, dev_Output_Soft_Back, fc_desc, HIPDNN_CONVOLUTION_BWD_FILTER_ALGO_0,
		dev_WS3, WS_size3, &beta, weights_desc, dev_Filter_Gradient);

	//CPU�� ��� ����
	hipMemcpy(FCBack, dev_Filter_Gradient, sizeof(float) * 3 * 16 * 32, hipMemcpyDeviceToHost);


	//***************************
	//**Pooling Backpropagation**
	//***************************

	//Pooling Backward ������� ���� �� GPU�� �޸� �Ҵ�
	float PoolingBack[1][3][32][32];
	float * dev_Pool_Back;
	hipMalloc((void**)&dev_Pool_Back, sizeof(float) * 32 * 32 * 3);

	//Filter�� �����ؿ���
	float cpy_Filter[3][3];

	for (int row = 0; row < 3; row++)
	{
		for (int col = 0; col < 3; col++)
		{
			cpy_Filter[col][row] = FCBack[0][0][col][row];
		}
	}

	//������ Filter Gradient�� GPU �޸𸮿� �Ҵ�
	float * dev_Filter_cpy;
	hipMalloc((void**)&dev_Filter_cpy, sizeof(float) * 9);
	hipMemcpy(dev_Filter_cpy, cpy_Filter, sizeof(float) * 9, hipMemcpyHostToDevice);

	//����ü ������� �ʱ�ȭ
	hipdnnTensorDescriptor_t pool_back_desc;
	hipdnnCreateTensorDescriptor(&pool_back_desc);
	hipdnnSetTensor4dDescriptor(pool_back_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, 1, 3, 32, 32);


	hipdnnPoolingBackward(cudnn, pool_desc, &alpha,
		out_pool_desc, dev_Output_Pool, /**/out_pool_desc, dev_Filter_cpy/**/, out_conv_desc, dev_Output_Conv,
		&beta, pool_back_desc, dev_Pool_Back);

	hipMemcpy(PoolingBack, dev_Pool_Back, sizeof(float) * 32 * 32 * 3, hipMemcpyDeviceToHost);



	//����
	hipFree(dev_Filter_cpy);

	//******************************
	//**Activation Backpropagation**
	//******************************


	//���߿� ����...
	/*
	float ActBack[1][1][6][6];
	float * dev_Act_Back;
	hipMalloc((void**)&dev_Act_Back, sizeof(float) * 6 * 6);

	hipdnnActivationBackward(cudnn, act_desc,
	&alpha, out_conv_desc, dev_Output_Act, pool_back_desc, dev_Pool_Back, out_pool_desc, dev_Output_Pool,
	&beta, out_conv_desc, dev_Act_Back);

	hipMemcpy(ActBack, dev_Act_Back, sizeof(float) * 6 * 6, hipMemcpyDeviceToHost);
	*/



	//*******************************
	//**Convolution Backpropagation**
	//*******************************

	float BackConv[1][3][16][16];
	float *dev_Back_Conv;
	hipMalloc((void**)&dev_Back_Conv, sizeof(float) * 1 * 3 * 16 * 16);

	/*
	hipdnnTensorDescriptor_t back_fc_filt;
	hipdnnCreateTensorDescriptor(&back_fc_filt);
	hipdnnSetTensor4dDescriptor(back_fc_filt, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, 1, 1, 6, 6);
	*/

	//out_conv_desc, dev_Output_Conv

	//Convolution BackProp ����
	hipdnnConvolutionBackwardFilter(
		cudnn, &alpha, in_desc, dev_Input, pool_back_desc, dev_Pool_Back,
		conv_desc, HIPDNN_CONVOLUTION_BWD_FILTER_ALGO_0,
		dev_WS2, WS_size2, &beta, filt_desc, dev_Back_Conv);

	hipMemcpy(BackConv, dev_Back_Conv, sizeof(float) * 1 * 3 * 16 * 16, hipMemcpyDeviceToHost);


	//********
	//**����**
	//********

	//����ü
	hipdnnDestroyTensorDescriptor(in_desc);
	hipdnnDestroyFilterDescriptor(filt_desc);
	hipdnnDestroyConvolutionDescriptor(conv_desc);
	hipdnnDestroyTensorDescriptor(out_conv_desc);
	hipdnnDestroyTensorDescriptor(out_pool_desc);
	hipdnnDestroyPoolingDescriptor(pool_desc);
	hipdnnDestroyTensorDescriptor(bias_desc);
	hipdnnDestroyActivationDescriptor(act_desc);
	hipdnnDestroyFilterDescriptor(weights_desc);
	hipdnnDestroyTensorDescriptor(out_fc_desc);
	hipdnnDestroyTensorDescriptor(dif_soft_desc);
	hipdnnDestroyTensorDescriptor(pool_back_desc);



	//GPU �޸�
	hipFree(dev_Filt);
	hipFree(dev_Input);
	hipFree(dev_WS);
	hipFree(dev_Output_Conv);
	hipFree(dev_Output_Act);
	hipFree(dev_Output_Pool);
	hipFree(dev_Bias);
	hipFree(dev_weights);
	hipFree(dev_WS2);
	hipFree(dev_Output_FC);
	hipFree(dev_Bias_FC);

	hipFree(dev_Filter_Gradient);
	hipFree(dev_Pool_Back);
	hipFree(dev_Output_Softmax);
	hipFree(dif_Soft_Back);
	hipFree(dev_WS3);
	//hipFree(dev_Act_Back);

	//************
	//**ȭ�����**
	//************

	//Input

	std::cout << "*******************" << std::endl << "**Input Data ����**"
		<< std::endl << "*******************" << std::endl << std::endl;


	std::cout << "Input" << std::endl << std::endl;

	for (int i = 0; i < FeatureHeight; i++)
	{
		for (int j = 0; j < FeatureWidth; j++)
		{
			std::cout << setw(3) << Input[0][0][i][j] << "  ";
		}
		std::cout << std::endl;
	}


	//Filter
	std::cout << std::endl << std::endl << "Filter" << std::endl << std::endl;

	for (int i = 0; i < filt_h; i++)
	{
		for (int j = 0; j < filt_w; j++)
		{
			std::cout << setw(3) << Filter[0][0][i][j] << "  ";
		}
		std::cout << std::endl;
	}

	//Fully Connected Weights

	std::cout << std::endl << std::endl << "Output1 Weights" << std::endl << std::endl;

	for (int i = 0; i < 16; i++)
	{
		for (int j = 0; j < 16; j++)
		{
			std::cout << setw(3) << Weights[0][0][i][j] << "  ";
		}
		std::cout << std::endl;
	}

	std::cout << std::endl;

	std::cout << std::endl << std::endl << "Output2 Weights" << std::endl << std::endl;

	for (int i = 0; i < 16; i++)
	{
		for (int j = 0; j < 16; j++)
		{
			std::cout << setw(3) << Weights[0][0][i][j] << "  ";
		}
		std::cout << std::endl;
	}

	std::cout << std::endl;



	//Convolution���

	std::cout << std::endl << std::endl << std::endl <<
		"************************" << std::endl << "**Feedforward ������**"
		<< std::endl << "************************";


	std::cout << std::endl << std::endl << "Convolution ���" << std::endl << std::endl;

	for (int i = 0; i < out_conv_h; i++)
	{
		for (int j = 0; j < out_conv_w; j++)
		{
			std::cout << setw(3) << Output_Conv[0][0][i][j] << "  ";
		}
		std::cout << std::endl;
	}

	


	//Bias�� ���

	std::cout << std::endl << std::endl << "Add Bias (bias : -10)" << std::endl << std::endl;

	for (int i = 0; i < FeatureHeight; i++)
	{
		for (int j = 0; j < FeatureWidth; j++)
		{
			std::cout << setw(3) << Output_Bias[0][0][i][j] << "  ";
		}
		std::cout << std::endl;
	}

	//Actavation Function ���

	std::cout << std::endl << std::endl << "Activation Function ���" << std::endl << std::endl;

	for (int i = 0; i < FeatureHeight; i++)
	{
		for (int j = 0; j < FeatureWidth; j++)
		{
			std::cout << setw(3) << Output_Activation[0][0][i][j] << "  ";
		}
		std::cout << std::endl;
	}


	//Pooling��� 

	std::cout << std::endl << std::endl << "Pooling ���" << std::endl << std::endl;

	for (int i = 0; i < out_pool_h; i++)
	{
		for (int j = 0; j < out_pool_w; j++)
		{
			std::cout << Output_Pool[0][0][i][j] << "  ";
		}
		std::cout << std::endl;
	}

	//���ͷ� ��ȯ 

	std::cout << std::endl << std::endl << "Fully Connected Vector" << std::endl << std::endl;

	for (int i = 0; i < out_pool_h; i++)
	{
		for (int j = 0; j < out_pool_w; j++)
		{
			std::cout << Output_Pool[0][0][i][j] << "  ";
		}
		std::cout << std::endl;
	}

	std::cout << std::endl;


	//Fully Connected���

	std::cout << std::endl << std::endl << "Fully Connected ���� ���" << std::endl << std::endl;

	std::cout << Output_FC[0][0][0][0] << "  " << Output_FC[0][0][0][1] << std::endl;


	//Fully Connected Bias���

	std::cout << std::endl << std::endl << "Fully Connected Bias" << std::endl << std::endl;

	std::cout << Output_FC_Bias[0][0][0][0] << "  " << Output_FC_Bias[0][0][0][1] << std::endl;


	//Softmax ���

	std::cout << std::endl << std::endl << "Softmax ���" << std::endl << std::endl;

	std::cout << OutSoft[0][0][0][0] << "  " << OutSoft[0][0][0][1] << std::endl;

	//���ϴ� ��� (One-hot Encoding)

	std::cout << std::endl << std::endl << "One-hot Encoding" << std::endl << std::endl;

	std::cout << "1" << "  " << "0" << std::endl;

	//Cross Entropy

	std::cout << std::endl << std::endl << "Cross Entropy ��" << std::endl << std::endl;

	std::cout << error;

	//********************
	//********************
	//**Back-Propagation**
	//********************
	//********************


	std::cout << std::endl << std::endl << std::endl << std::endl <<
		"************************************" << std::endl << "**Backpropagation ���(delta) ���**"
		<< std::endl << "************************************" << std::endl << std::endl
		<< "�н��� : 0.01" << std::endl;


	//Softmax BackProp - delta

	std::cout << std::endl << std::endl << "Diff Softmax - delta Softmax (dZ)" << std::endl << std::endl;

	std::cout << SoftBack[0][0][0][0] << "  " << SoftBack[0][0][0][1] << std::endl;




	//BackProp of Fully Connected Bias

	/*

	std::cout << std::endl << std::endl << "FC Bias Backward - delta bias of FC" << std::endl << std::endl;

	std::cout << FCbiasBack[0][0][0][0] << std::endl;

	*/


	//BackProp of Fully Connected ���

	std::cout << std::endl << std::endl << "Output1 - delta Weight1 (dW1)" << std::endl << std::endl;

	for (int i = 0; i < 3; i++)
	{
		for (int j = 0; j < 3; j++)
		{
			std::cout  << FCBack[0][0][i][j] << "  ";
		}
		std::cout << std::endl;
	}

	std::cout << std::endl << std::endl << std::endl << "Output2 - delta Weight2 (cW2)" << std::endl << std::endl;

	for (int i = 0; i < 3; i++)
	{
		for (int j = 3; j < 6; j++)
		{
			std::cout << FCBack[0][0][i][j] << "  ";
		}
		std::cout << std::endl;
	}

	std::cout << std::endl;


	//BakProp of Pooling

	std::cout << std::endl << std::endl << "Pool Back - delta Pooling Filter" << std::endl << std::endl;

	for (int i = 0; i < FeatureHeight; i++)
	{
		for (int j = 0; j < FeatureWidth; j++)
		{
			std::cout << PoolingBack[0][0][i][j] << "  ";
		}
		std::cout << std::endl;
	}



	//BackProp of Activation Layer

	/*
	std::cout << std::endl << std::endl << "Act Back - delta Activaton Function" << std::endl << std::endl;

	for (int i = 0; i < FeatureHeight; i++)
	{
	for (int j = 0; j < FeatureWidth; j++)
	{
	std::cout << ActBack[0][0][i][j] << "  ";
	}
	std::cout << std::endl;
	}
	*/

	//BackProp of Convolution

	std::cout << std::endl << std::endl << "Convolution Back - delta Filter (dW)" << std::endl << std::endl;

	for (int i = 0; i < 3; i++)
	{
		for (int j = 0; j < 3; j++)
		{
			std::cout << BackConv[0][0][i][j] << "  ";
		}
		std::cout << std::endl;
	}



	std::cout << std::endl << std::endl;


	
}
