#include "hip/hip_runtime.h"
#include <math.h>
#include <iostream>
#include <iomanip>
#include <hip/hip_runtime.h>
#include <hipDNN.h>
#include <hip/hip_runtime.h>
#include <cstdlib>
#include <cassert>
#include <io.h>
#include <vector>
#include <random>
#include <hipblas.h>

#include <stdio.h>
#include <string.h>
#include <opencv2/opencv.hpp>
#include <>

#define BW 512

using namespace std;
using namespace cv;


__global__ void MomentumInitialize(
	const size_t NextLayerNodeNumber/*Number of output  feature maps*/,
	const size_t PrevLayerNodeNumber/*Number of input feature maps*/,
	const size_t Height/*Height of each filter.*/,
	const size_t Width/*Width of each filter.*/, float *diff)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx >= NextLayerNodeNumber * PrevLayerNodeNumber * Height * Width)
		return;

	diff[idx] = 0.0f;
}

__global__ void SoftmaxLossBackprop(const float *label, const int num_labels, const int batch_size, float *diff)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx >= batch_size)
		return;

	const int label_value = static_cast<int>(label[idx]);

	// For each item in the batch, decrease the result of the label's value by 1
	diff[idx * num_labels + label_value] -= 1.0f;
}

vector<pair<Mat, string>> TraverseFilesUsingDFS(const string& folder_path)
{
	_finddata_t file_info;
	string any_file_pattern = folder_path + "\\*";
	intptr_t handle = _findfirst(any_file_pattern.c_str(), &file_info);
	vector<pair<Mat, string>> ImgBox;

	//If folder_path exsist, using any_file_pattern will find at least two files "." and "..",
	//of which "." means current dir and ".." means parent dir
	if (handle == -1)
	{
		cerr << "folder path not exist: " << folder_path << endl;
		exit(-1);
	}

	//iteratively check each file or sub_directory in current folder
	do
	{
		string file_name = file_info.name; //from char array to string

										   //check whtether it is a sub direcotry or a file
		if (file_info.attrib & _A_SUBDIR)
		{
			if (file_name != "." && file_name != "..")
			{
				string sub_folder_path = folder_path + "\\" + file_name;
				TraverseFilesUsingDFS(sub_folder_path);
				cout << "a sub_folder path: " << sub_folder_path << endl;
			}
		}
		else  //cout << "file name: " << file_name << endl;
		{
			size_t npo1 = file_name.find('_') + 1;
			size_t npo2 = file_name.find('.');
			size_t npo3 = npo2 - npo1;
			string newname = file_name.substr(npo1, npo3);
			string sub_folder_path2 = folder_path + "\\" + file_name;
			Mat img = imread(sub_folder_path2);
			ImgBox.push_back({ { img },{ newname } });
		}
	} while (_findnext(handle, &file_info) == 0);

	//
	_findclose(handle);
	return ImgBox;
}

void checkCUDNN(hipdnnStatus_t status)
{
	if (status != HIPDNN_STATUS_SUCCESS)
		std::cout << "[ERROR] CUDNN " << status << std::endl;
}

void checkCudaErrors(hipError_t error)
{
	if (error != hipSuccess)
		std::cout << "[ERROR] CUDA " << error << std::endl;
}

void InitWeightsXavier(float* Weights,
	const size_t NextLayerNodeNumber/*Number of output  feature maps*/,
	const size_t PrevLayerNodeNumber/*Number of input feature maps*/,
	const size_t Height/*Height of each filter.*/,
	const size_t Width/*Width of each filter.*/)
{

	random_device rd;
	mt19937 gen(rd());
	float sigma = sqrt(6.0f / static_cast<float>((NextLayerNodeNumber + PrevLayerNodeNumber) * Height * Width));
	uniform_real_distribution<float> d(-sigma, sigma);

	//Weights ����
	for (int och = 0; och < NextLayerNodeNumber; och++)
	{
		for (int ch = 0; ch < PrevLayerNodeNumber; ch++)
		{
			for (int row = 0; row < Height; row++)
			{
				for (int col = 0; col < Width; col++)
				{
					Weights[och * PrevLayerNodeNumber * Height * Width + ch *  Height * Width + row * Width + col] = static_cast<float>(d(gen));
				}
			}
		}
	}

}


void InitWeightsbias(float* Weightsbias, const size_t & numOutSize)
{
	for (int i = 0; i < numOutSize; i++)
	{
		Weightsbias[i] = 0.0f;
	}
}






int main()
{
	time_t startTime = 0, endTime = 0;
	time_t startTime_train = 0, endTime_train = 0;

	startTime = clock();

	const int num_labels = 10; // �� ��

	int ImageNum = 50000; // �̹��� �� ����
	int batchSize = 100; // Ʈ���̴� ��ġ ��

	int ImageNum_test = 10000; // �̹��� �� ����
	const int batch_size_test = 100; // �׽�Ʈ ��ġ ��

	int epoch = 100; // epoch Ƚ�� 

	//Learning Rate
	//float learning_rate = -0.001;
	//Momentum
	float learning_rate = 0.0001;
	float momentum = 0.9;
	float eta = -1.0;


	int input_channelCnt = 3, imageHeight = 32, imageWidth = 32;
	//int input_channelCnt = 1, imageHeight = 28, imageWidth = 28;


	vector<pair<Mat, string>> ImgBox; // �̹��� ������, �̹��� �̸�
	ImgBox = TraverseFilesUsingDFS("C:\\cifar\\train");// �̹����� ����Ǿ� �ִ� ���� ���
													   //ImgBox = TraverseFilesUsingDFS("C:\\Users\\ECMUser\\Desktop\\DataSet\\MNIST11000\\trainset");// �̹����� ����Ǿ� �ִ� ���� ���
	vector<string> LabelBox; // �� ������ ���� ����
	vector<pair<int, string>> LabelTable; // �󺧸� ���� �ѹ� �ο�

	vector<pair<Mat, string>> ImgBox_test; // �̹��� ������, �̹��� �̸�
	ImgBox_test = TraverseFilesUsingDFS("C:\\cifar\\test");// �̹����� ����Ǿ� �ִ� ���� ���
														   //ImgBox_test = TraverseFilesUsingDFS("C:\\Users\\ECMUser\\Desktop\\DataSet\\cifar_10\\test");// �̹����� ����Ǿ� �ִ� ���� ���
														   //ImgBox_test = TraverseFilesUsingDFS("C:\\Users\\ECMUser\\Desktop\\DataSet\\MNIST11000\\testset");// �̹����� ����Ǿ� �ִ� ���� ���

	float* target_train = new float[ImageNum]; // target �� , �󺧿� ���� ������ �ѹ� ���� ��� �迭 

											   // �󺧿� ��ȣ �ο��� ���� LabelBox ���Ϳ� �� ���� �ϰ� ���� �� �ߺ� ����
	for (int i = 0; i < ImageNum; i++)
	{
		//std::cout<< "�� ��� :: " << ImgBox[i].second << std::endl; // �Է¹���������� �� ��� -> ���� "�� ��� :: automobile"
		LabelBox.push_back(ImgBox[i].second);
	}

	sort(LabelBox.begin(), LabelBox.end());
	LabelBox.erase(unique(LabelBox.begin(), LabelBox.end()), LabelBox.end());
	int nLabelBoxSize = LabelBox.size();

	// �� ��ȣ �ο�
	for (int i = 0; i < nLabelBoxSize; i++)
	{
		LabelTable.push_back({ { i },{ LabelBox[i] } });
		//std::cout << "LabelBox :: " << LabelBox[i] << std::endl;// -> ���� "LabelBox :: truck"
	}

	//target ����
	for (int i = 0; i < ImageNum; i++) {
		for (int j = 0; j < LabelTable.size(); j++) {
			if (ImgBox[i].second == LabelTable[j].second) {
				target_train[i] = LabelTable[j].first;
			}
		}
	}


	cout << "=================== �̹��� �ҷ����� ���� ====================" << endl;
	endTime = clock();
	printf("�̹��� �غ� �ð�: %.1f �� \n", (float)(endTime - startTime) / ((CLOCKS_PER_SEC) * 60));

	hipdnnHandle_t cudnn;
	hipdnnCreate(&cudnn);
	hipblasHandle_t cublasHandle;
	hipblasCreate(&cublasHandle);

	float alpha = 1.0;
	float beta = 0.0;


	float* Input_train = new float[batchSize * input_channelCnt * imageHeight * imageWidth];
	float* target_train_batch = new float[batchSize];


	const int conv1FilterCnt = 6, conv1FilterHeight = 5, conv1FilterWidth = 5;
	int conv1Pad_h = 0, conv1Pad_w = 0, conv1Str_h = 1, conv1Str_w = 1, conv1Dil_h = 1, conv1Dil_w = 1;

	int conv2FilterCnt = 16, conv2FilterHeight = 5, conv2FilterWidth = 5;
	int conv2Pad_h = 0, conv2Pad_w = 0, conv2Str_h = 1, conv2Str_w = 1, conv2Dil_h = 1, conv2Dil_w = 1;
	//int conv2Pad_h = 2, conv2Pad_w = 2, conv2Str_h = 1, conv2Str_w = 1, conv2Dil_h = 1, conv2Dil_w = 1;

	int poolWind_h = 2, poolWind_w = 2, poolPad_h = 0, poolPad_w = 0, poolStrd_w = 2, poolStrd_h = 2; //MAX POOLING ����

	int fcPad_h = 0, fcPad_w = 0, fcStr_h = 1, fcStr_w = 1, fcDil_h = 1, fcDil_w = 1; //�굵 ����

	int lastLayer_numOut = 10; // ��� Ŭ���� ��


	int conv1OutHeight = 28;
	int conv1OutWidth = 28;

	int pool1OutHeight = 14;
	int pool1OutWidth = 14;

	int conv2OutHeight = 10;
	int conv2OutWidth = 10;

	int pool2OutHeight = 5;
	int pool2OutWidth = 5;

	int fc1FilterCnt = 120;
	int fc2FilterCnt = 84;



	//Weight initialization

	//conv1 filter
	float* conv1Filter = new float[conv1FilterCnt * input_channelCnt * conv1FilterHeight * conv1FilterWidth];
	InitWeightsXavier(conv1Filter, conv1FilterCnt, input_channelCnt, conv1FilterHeight, conv1FilterWidth);

	//conv2 filter
	float* conv2Filter = new float[conv2FilterCnt * conv1FilterCnt * conv2FilterHeight * conv2FilterWidth];
	InitWeightsXavier(conv2Filter, conv2FilterCnt, conv1FilterCnt, conv2FilterHeight, conv2FilterWidth);

	//fc1 filter
	float* fc1Filter = new float[fc1FilterCnt * conv2FilterCnt * pool2OutHeight * pool2OutWidth];
	InitWeightsXavier(fc1Filter, fc1FilterCnt, conv2FilterCnt, pool2OutHeight, pool2OutWidth);

	//fc2 filter
	float* fc2Filter = new float[fc2FilterCnt * fc1FilterCnt * 1 * 1];
	InitWeightsXavier(fc2Filter, fc2FilterCnt, fc1FilterCnt, 1, 1);

	// fc3 filter
	float* fc3Filter = new float[lastLayer_numOut * fc2FilterCnt * 1 * 1];
	InitWeightsXavier(fc3Filter, lastLayer_numOut, fc2FilterCnt, 1, 1);

	//conv1 bias
	float* conv1Bias = new float[conv1FilterCnt];
	InitWeightsbias(conv1Bias, conv1FilterCnt);

	// conv2 bias
	float* conv2Bias = new float[conv2FilterCnt];
	InitWeightsbias(conv2Bias, conv2FilterCnt);

	//fc1 bias
	float* fc1Bias = new float[fc1FilterCnt];
	InitWeightsbias(fc1Bias, fc1FilterCnt);

	// fc2 bias
	float* fc2Bias = new float[fc2FilterCnt];
	InitWeightsbias(fc2Bias, fc2FilterCnt);

	// fc3 bias
	float* fc3Bias = new float[lastLayer_numOut];
	InitWeightsbias(fc3Bias, lastLayer_numOut);






	float* dev_conv1_Filter;
	checkCudaErrors(hipMalloc((void**)&dev_conv1_Filter, sizeof(float) * conv1FilterCnt * input_channelCnt * conv1FilterHeight * conv1FilterWidth));
	checkCudaErrors(hipMemcpy(dev_conv1_Filter, conv1Filter, sizeof(float) * conv1FilterCnt * input_channelCnt * conv1FilterHeight * conv1FilterWidth, hipMemcpyHostToDevice));

	float* dev_conv1_Output;
	checkCudaErrors(hipMalloc((void**)&dev_conv1_Output, sizeof(float) * batchSize * conv1FilterCnt * conv1OutHeight * conv1OutWidth));

	float* dev_conv1_Bias;
	checkCudaErrors(hipMalloc((void**)&dev_conv1_Bias, sizeof(float) * 1 * conv1FilterCnt * 1 * 1));
	checkCudaErrors(hipMemcpy(dev_conv1_Bias, conv1Bias, sizeof(float) * 1 * conv1FilterCnt * 1 * 1, hipMemcpyHostToDevice));

	float* dev_conv1Act_output;
	checkCudaErrors(hipMalloc((void**)&dev_conv1Act_output, sizeof(float) * batchSize * conv1FilterCnt * conv1OutHeight * conv1OutWidth));

	float* dev_pool1_Output;
	checkCudaErrors(hipMalloc((void**)&dev_pool1_Output, sizeof(float) * batchSize * conv1FilterCnt * pool1OutHeight * pool1OutWidth));

	float* dev_conv1pool_Act_output;
	checkCudaErrors(hipMalloc((void**)&dev_conv1pool_Act_output, sizeof(float) * batchSize * conv1FilterCnt * pool1OutHeight * pool1OutWidth));

	float* dev_conv2_Filter;
	checkCudaErrors(hipMalloc((void**)&dev_conv2_Filter, sizeof(float) * conv2FilterCnt * conv1FilterCnt * conv2FilterHeight * conv2FilterWidth));
	checkCudaErrors(hipMemcpy(dev_conv2_Filter, conv2Filter, sizeof(float) * conv2FilterCnt * conv1FilterCnt * conv2FilterHeight * conv2FilterWidth, hipMemcpyHostToDevice));

	float* dev_conv2_Output;
	checkCudaErrors(hipMalloc((void**)&dev_conv2_Output, sizeof(float) * batchSize * conv2FilterCnt * conv2OutHeight * conv2OutWidth));

	float* dev_conv2_Bias;
	checkCudaErrors(hipMalloc((void**)&dev_conv2_Bias, sizeof(float) * 1 * conv2FilterCnt * 1 * 1));
	checkCudaErrors(hipMemcpy(dev_conv2_Bias, conv2Bias, sizeof(float) * 1 * conv2FilterCnt * 1 * 1, hipMemcpyHostToDevice));

	float* dev_conv2Act_output;
	checkCudaErrors(hipMalloc((void**)&dev_conv2Act_output, sizeof(float) * batchSize * conv2FilterCnt * conv2OutHeight * conv2OutWidth));

	float* dev_pool2_Output;
	checkCudaErrors(hipMalloc((void**)&dev_pool2_Output, sizeof(float) * batchSize * conv2FilterCnt * pool2OutHeight * pool2OutWidth));

	float* dev_conv2pool_Act_output;
	checkCudaErrors(hipMalloc((void**)&dev_conv2pool_Act_output, sizeof(float) * batchSize * conv2FilterCnt * pool2OutHeight * pool2OutWidth));

	float* dev_fc1_Filter;
	checkCudaErrors(hipMalloc((void**)&dev_fc1_Filter, sizeof(float) * fc1FilterCnt * conv2FilterCnt * pool2OutHeight * pool2OutWidth));
	checkCudaErrors(hipMemcpy(dev_fc1_Filter, fc1Filter, sizeof(float) * fc1FilterCnt * conv2FilterCnt * pool2OutHeight * pool2OutWidth, hipMemcpyHostToDevice));

	float* dev_fc1_Output;
	checkCudaErrors(hipMalloc((void**)&dev_fc1_Output, sizeof(float) * batchSize * fc1FilterCnt * 1 * 1));

	float* dev_fc1_Bias;
	checkCudaErrors(hipMalloc((void**)&dev_fc1_Bias, sizeof(float) * 1 * fc1FilterCnt * 1 * 1));
	checkCudaErrors(hipMemcpy(dev_fc1_Bias, fc1Bias, sizeof(float) * 1 * fc1FilterCnt * 1 * 1, hipMemcpyHostToDevice));

	float* dev_fc1_Actout;
	checkCudaErrors(hipMalloc((void**)&dev_fc1_Actout, sizeof(float) * batchSize * fc1FilterCnt * 1 * 1));

	float* dev_fc2_Filter;
	checkCudaErrors(hipMalloc((void**)&dev_fc2_Filter, sizeof(float) * fc2FilterCnt * fc1FilterCnt * 1 * 1));
	checkCudaErrors(hipMemcpy(dev_fc2_Filter, fc2Filter, sizeof(float) * fc2FilterCnt * fc1FilterCnt * 1 * 1, hipMemcpyHostToDevice));

	float* dev_fc2_Output;
	checkCudaErrors(hipMalloc((void**)&dev_fc2_Output, sizeof(float) * batchSize * fc2FilterCnt * 1 * 1));

	float* dev_fc2_Bias;
	checkCudaErrors(hipMalloc((void**)&dev_fc2_Bias, sizeof(float) * 1 * fc2FilterCnt * 1 * 1));
	checkCudaErrors(hipMemcpy(dev_fc2_Bias, fc2Bias, sizeof(float) * 1 * fc2FilterCnt * 1 * 1, hipMemcpyHostToDevice));

	float* dev_fc2_Actout;
	checkCudaErrors(hipMalloc((void**)&dev_fc2_Actout, sizeof(float) * batchSize * fc2FilterCnt * 1 * 1));

	float* dev_fc3_Filter;
	checkCudaErrors(hipMalloc((void**)&dev_fc3_Filter, sizeof(float) * lastLayer_numOut * fc2FilterCnt * 1 * 1));
	checkCudaErrors(hipMemcpy(dev_fc3_Filter, fc3Filter, sizeof(float) * lastLayer_numOut * fc2FilterCnt * 1 * 1, hipMemcpyHostToDevice));

	float* dev_fc3_Output;
	checkCudaErrors(hipMalloc((void**)&dev_fc3_Output, sizeof(float) * batchSize * lastLayer_numOut * 1 * 1));

	float* dev_fc3_Bias;
	checkCudaErrors(hipMalloc((void**)&dev_fc3_Bias, sizeof(float) * 1 * lastLayer_numOut * 1 * 1));
	checkCudaErrors(hipMemcpy(dev_fc3_Bias, fc3Bias, sizeof(float) * 1 * lastLayer_numOut * 1 * 1, hipMemcpyHostToDevice));

	float* dev_smaxOutput;
	checkCudaErrors(hipMalloc((void**)&dev_smaxOutput, sizeof(float) * batchSize * lastLayer_numOut * 1 * 1));



	/**********FEEDFORWARD********/

	//hipdnnPoolingMode_t poolMode = HIPDNN_POOLING_AVERAGE_COUNT_EXCLUDE_PADDING;
	hipdnnPoolingMode_t poolMode = HIPDNN_POOLING_MAX;

	//hipdnnActivationMode_t fwActalgo = HIPDNN_ACTIVATION_RELU;
	hipdnnActivationMode_t fwActalgo = HIPDNN_ACTIVATION_TANH;
	//hipdnnActivationMode_t fwActalgo = HIPDNN_ACTIVATION_SIGMOID;

	//hipdnnSoftmaxAlgorithm_t sftAlgo = HIPDNN_SOFTMAX_FAST;
	hipdnnSoftmaxAlgorithm_t sftAlgo = HIPDNN_SOFTMAX_ACCURATE;
	hipdnnSoftmaxMode_t sftMode = HIPDNN_SOFTMAX_MODE_INSTANCE;
	//hipdnnSoftmaxMode_t sftMode = HIPDNN_SOFTMAX_MODE_CHANNEL;


	hipdnnTensorDescriptor_t input_Tensor;
	checkCUDNN(hipdnnCreateTensorDescriptor(&input_Tensor));
	checkCUDNN(hipdnnSetTensor4dDescriptor(input_Tensor, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, batchSize, input_channelCnt, imageHeight, imageWidth));

	hipdnnFilterDescriptor_t conv1_Filter_Desc;
	checkCUDNN(hipdnnCreateFilterDescriptor(&conv1_Filter_Desc));
	checkCUDNN(hipdnnSetFilter4dDescriptor(conv1_Filter_Desc, HIPDNN_DATA_FLOAT, HIPDNN_TENSOR_NCHW, conv1FilterCnt, input_channelCnt, conv1FilterHeight, conv1FilterWidth));

	hipdnnConvolutionDescriptor_t conv1_Desc;
	checkCUDNN(hipdnnCreateConvolutionDescriptor(&conv1_Desc));
	checkCUDNN(hipdnnSetConvolution2dDescriptor(conv1_Desc, conv1Pad_h, conv1Pad_w, conv1Str_h, conv1Str_w, conv1Dil_h, conv1Dil_w, HIPDNN_CROSS_CORRELATION, HIPDNN_DATA_FLOAT));

	hipdnnTensorDescriptor_t conv1_Tensor;
	checkCUDNN(hipdnnCreateTensorDescriptor(&conv1_Tensor));
	int conv1Out_n, conv1Out_c, conv1Out_h, conv1Out_w;
	checkCUDNN(hipdnnGetConvolution2dForwardOutputDim(conv1_Desc, input_Tensor, conv1_Filter_Desc, &conv1Out_n, &conv1Out_c, &conv1Out_h, &conv1Out_w));
	checkCUDNN(hipdnnSetTensor4dDescriptor(conv1_Tensor, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, conv1Out_n, conv1Out_c, conv1Out_h, conv1Out_w));

	hipdnnConvolutionFwdAlgo_t conv1_fwAlgo;
	checkCUDNN(hipdnnGetConvolutionForwardAlgorithm(cudnn, input_Tensor, conv1_Filter_Desc, conv1_Desc, conv1_Tensor, HIPDNN_CONVOLUTION_FWD_PREFER_FASTEST, 0, &conv1_fwAlgo));
	size_t conv1_worksSize = 0;
	checkCUDNN(hipdnnGetConvolutionForwardWorkspaceSize(cudnn, input_Tensor, conv1_Filter_Desc, conv1_Desc, conv1_Tensor, conv1_fwAlgo, &conv1_worksSize));
	size_t * dev_conv1_works;
	checkCudaErrors(hipMalloc((void**)&dev_conv1_works, conv1_worksSize));

	hipdnnTensorDescriptor_t conv1_biasTensor;
	checkCUDNN(hipdnnCreateTensorDescriptor(&conv1_biasTensor));
	checkCUDNN(hipdnnSetTensor4dDescriptor(conv1_biasTensor, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, 1, conv1FilterCnt, 1, 1));

	hipdnnActivationDescriptor_t conv1_Act_Desc;
	checkCUDNN(hipdnnCreateActivationDescriptor(&conv1_Act_Desc));
	checkCUDNN(hipdnnSetActivationDescriptor(conv1_Act_Desc, fwActalgo, HIPDNN_PROPAGATE_NAN, 0));

	hipdnnPoolingDescriptor_t pool1_Desc;
	checkCUDNN(hipdnnCreatePoolingDescriptor(&pool1_Desc));
	checkCUDNN(hipdnnSetPooling2dDescriptor(pool1_Desc, poolMode, HIPDNN_PROPAGATE_NAN, poolWind_h, poolWind_w, poolPad_h, poolPad_w, poolStrd_h, poolStrd_w));

	hipdnnTensorDescriptor_t pool1_Tensor;
	checkCUDNN(hipdnnCreateTensorDescriptor(&pool1_Tensor));
	int pool1Out_n, pool1Out_c, pool1Out_h, pool1Out_w;
	checkCUDNN(hipdnnGetPooling2dForwardOutputDim(pool1_Desc, conv1_Tensor, &pool1Out_n, &pool1Out_c, &pool1Out_h, &pool1Out_w));
	checkCUDNN(hipdnnSetTensor4dDescriptor(pool1_Tensor, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, pool1Out_n, pool1Out_c, pool1Out_h, pool1Out_w));

	hipdnnActivationDescriptor_t conv1pool_Act_Desc;
	checkCUDNN(hipdnnCreateActivationDescriptor(&conv1pool_Act_Desc));
	checkCUDNN(hipdnnSetActivationDescriptor(conv1pool_Act_Desc, fwActalgo, HIPDNN_PROPAGATE_NAN, 0));

	hipdnnFilterDescriptor_t conv2_Filter_Desc;
	checkCUDNN(hipdnnCreateFilterDescriptor(&conv2_Filter_Desc));
	checkCUDNN(hipdnnSetFilter4dDescriptor(conv2_Filter_Desc, HIPDNN_DATA_FLOAT, HIPDNN_TENSOR_NCHW, conv2FilterCnt, conv1FilterCnt, conv2FilterHeight, conv2FilterWidth));

	hipdnnConvolutionDescriptor_t conv2_Desc;
	checkCUDNN(hipdnnCreateConvolutionDescriptor(&conv2_Desc));
	checkCUDNN(hipdnnSetConvolution2dDescriptor(conv2_Desc, conv2Pad_h, conv2Pad_w, conv2Str_h, conv2Str_w, conv2Dil_h, conv2Dil_w, HIPDNN_CROSS_CORRELATION, HIPDNN_DATA_FLOAT));

	hipdnnTensorDescriptor_t conv2_Tensor;
	checkCUDNN(hipdnnCreateTensorDescriptor(&conv2_Tensor));
	int conv2Out_n, conv2Out_c, conv2Out_h, conv2Out_w;
	checkCUDNN(hipdnnGetConvolution2dForwardOutputDim(conv2_Desc, pool1_Tensor, conv2_Filter_Desc, &conv2Out_n, &conv2Out_c, &conv2Out_h, &conv2Out_w));
	checkCUDNN(hipdnnSetTensor4dDescriptor(conv2_Tensor, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, conv2Out_n, conv2Out_c, conv2Out_h, conv2Out_w));

	hipdnnConvolutionFwdAlgo_t conv2_fwAlgo;
	checkCUDNN(hipdnnGetConvolutionForwardAlgorithm(cudnn, pool1_Tensor, conv2_Filter_Desc, conv2_Desc, conv2_Tensor, HIPDNN_CONVOLUTION_FWD_PREFER_FASTEST, 0, &conv2_fwAlgo));
	size_t conv2_worksSize = 0;
	checkCUDNN(hipdnnGetConvolutionForwardWorkspaceSize(cudnn, pool1_Tensor, conv2_Filter_Desc, conv2_Desc, conv2_Tensor, conv2_fwAlgo, &conv2_worksSize));
	size_t * dev_conv2_works;
	checkCudaErrors(hipMalloc((void**)&dev_conv2_works, conv2_worksSize));

	hipdnnTensorDescriptor_t conv2_biasTensor;
	checkCUDNN(hipdnnCreateTensorDescriptor(&conv2_biasTensor));
	checkCUDNN(hipdnnSetTensor4dDescriptor(conv2_biasTensor, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, 1, conv2FilterCnt, 1, 1));

	hipdnnActivationDescriptor_t conv2_Act_Desc;
	checkCUDNN(hipdnnCreateActivationDescriptor(&conv2_Act_Desc));
	checkCUDNN(hipdnnSetActivationDescriptor(conv2_Act_Desc, fwActalgo, HIPDNN_PROPAGATE_NAN, 0));

	hipdnnPoolingDescriptor_t pool2_Desc;
	checkCUDNN(hipdnnCreatePoolingDescriptor(&pool2_Desc));
	checkCUDNN(hipdnnSetPooling2dDescriptor(pool2_Desc, poolMode, HIPDNN_PROPAGATE_NAN, poolWind_h, poolWind_w, poolPad_h, poolPad_w, poolStrd_h, poolStrd_w));

	hipdnnTensorDescriptor_t pool2_Tensor;
	checkCUDNN(hipdnnCreateTensorDescriptor(&pool2_Tensor));
	int pool2Out_n, pool2Out_c, pool2Out_h, pool2Out_w;
	checkCUDNN(hipdnnGetPooling2dForwardOutputDim(pool2_Desc, conv2_Tensor, &pool2Out_n, &pool2Out_c, &pool2Out_h, &pool2Out_w));
	checkCUDNN(hipdnnSetTensor4dDescriptor(pool2_Tensor, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, pool2Out_n, pool2Out_c, pool2Out_h, pool2Out_w));

	hipdnnActivationDescriptor_t conv2pool_Act_Desc;
	checkCUDNN(hipdnnCreateActivationDescriptor(&conv2pool_Act_Desc));
	checkCUDNN(hipdnnSetActivationDescriptor(conv2pool_Act_Desc, fwActalgo, HIPDNN_PROPAGATE_NAN, 0));

	hipdnnFilterDescriptor_t fc1_Filter_Desc;
	checkCUDNN(hipdnnCreateFilterDescriptor(&fc1_Filter_Desc));
	checkCUDNN(hipdnnSetFilter4dDescriptor(fc1_Filter_Desc, HIPDNN_DATA_FLOAT, HIPDNN_TENSOR_NCHW, fc1FilterCnt, conv2FilterCnt, pool2OutHeight, pool2OutWidth));

	hipdnnConvolutionDescriptor_t fc1_Desc;
	checkCUDNN(hipdnnCreateConvolutionDescriptor(&fc1_Desc));
	checkCUDNN(hipdnnSetConvolution2dDescriptor(fc1_Desc, fcPad_h, fcPad_w, fcStr_h, fcStr_w, fcDil_h, fcDil_w, HIPDNN_CROSS_CORRELATION, HIPDNN_DATA_FLOAT));

	hipdnnTensorDescriptor_t fc1_Tensor;
	checkCUDNN(hipdnnCreateTensorDescriptor(&fc1_Tensor));
	int fwd1Out_n, fwd1Out_c, fwd1Out_h, fwd1Out_w;
	checkCUDNN(hipdnnGetConvolution2dForwardOutputDim(fc1_Desc, pool2_Tensor, fc1_Filter_Desc, &fwd1Out_n, &fwd1Out_c, &fwd1Out_h, &fwd1Out_w));
	checkCUDNN(hipdnnSetTensor4dDescriptor(fc1_Tensor, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, fwd1Out_n, fwd1Out_c, fwd1Out_h, fwd1Out_w));

	hipdnnConvolutionFwdAlgo_t fc1_fwAlgo;
	checkCUDNN(hipdnnGetConvolutionForwardAlgorithm(cudnn, pool2_Tensor, fc1_Filter_Desc, fc1_Desc, fc1_Tensor, HIPDNN_CONVOLUTION_FWD_PREFER_FASTEST, 0, &fc1_fwAlgo));
	size_t fc1_worksSize = 0;
	checkCUDNN(hipdnnGetConvolutionForwardWorkspaceSize(cudnn, pool2_Tensor, fc1_Filter_Desc, fc1_Desc, fc1_Tensor, fc1_fwAlgo, &fc1_worksSize));
	size_t * dev_fc1_works;
	checkCudaErrors(hipMalloc((void**)&dev_fc1_works, fc1_worksSize));

	hipdnnTensorDescriptor_t fc1_biasTensor;
	checkCUDNN(hipdnnCreateTensorDescriptor(&fc1_biasTensor));
	checkCUDNN(hipdnnSetTensor4dDescriptor(fc1_biasTensor, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, 1, fc1FilterCnt, 1, 1));

	hipdnnActivationDescriptor_t fc1_Act_Desc;
	checkCUDNN(hipdnnCreateActivationDescriptor(&fc1_Act_Desc));
	checkCUDNN(hipdnnSetActivationDescriptor(fc1_Act_Desc, fwActalgo, HIPDNN_PROPAGATE_NAN, 0));

	hipdnnFilterDescriptor_t fc2_Filter_Desc;
	checkCUDNN(hipdnnCreateFilterDescriptor(&fc2_Filter_Desc));
	checkCUDNN(hipdnnSetFilter4dDescriptor(fc2_Filter_Desc, HIPDNN_DATA_FLOAT, HIPDNN_TENSOR_NCHW, fc2FilterCnt, fc1FilterCnt, 1, 1));

	hipdnnConvolutionDescriptor_t fc2_Desc;
	checkCUDNN(hipdnnCreateConvolutionDescriptor(&fc2_Desc));
	checkCUDNN(hipdnnSetConvolution2dDescriptor(fc2_Desc, fcPad_h, fcPad_w, fcStr_h, fcStr_w, fcDil_h, fcDil_w, HIPDNN_CROSS_CORRELATION, HIPDNN_DATA_FLOAT));

	hipdnnTensorDescriptor_t fc2_Tensor;
	checkCUDNN(hipdnnCreateTensorDescriptor(&fc2_Tensor));
	int fwd2Out_n, fwd2Out_c, fwd2Out_h, fwd2Out_w;
	checkCUDNN(hipdnnGetConvolution2dForwardOutputDim(fc2_Desc, fc1_Tensor, fc2_Filter_Desc, &fwd2Out_n, &fwd2Out_c, &fwd2Out_h, &fwd2Out_w));
	checkCUDNN(hipdnnSetTensor4dDescriptor(fc2_Tensor, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, fwd2Out_n, fwd2Out_c, fwd2Out_h, fwd2Out_w));

	hipdnnConvolutionFwdAlgo_t fc2_fwAlgo;
	checkCUDNN(hipdnnGetConvolutionForwardAlgorithm(cudnn, fc1_Tensor, fc2_Filter_Desc, fc2_Desc, fc2_Tensor, HIPDNN_CONVOLUTION_FWD_PREFER_FASTEST, 0, &fc2_fwAlgo));
	size_t fc2_worksSize = 0;
	checkCUDNN(hipdnnGetConvolutionForwardWorkspaceSize(cudnn, fc1_Tensor, fc2_Filter_Desc, fc2_Desc, fc2_Tensor, fc2_fwAlgo, &fc2_worksSize));
	size_t * dev_fc2_works;
	checkCudaErrors(hipMalloc((void**)&dev_fc2_works, fc2_worksSize));

	hipdnnTensorDescriptor_t fc2_biasTensor;
	checkCUDNN(hipdnnCreateTensorDescriptor(&fc2_biasTensor));
	checkCUDNN(hipdnnSetTensor4dDescriptor(fc2_biasTensor, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, 1, fc2FilterCnt, 1, 1));

	hipdnnActivationDescriptor_t fc2_Act_Desc;
	checkCUDNN(hipdnnCreateActivationDescriptor(&fc2_Act_Desc));
	checkCUDNN(hipdnnSetActivationDescriptor(fc2_Act_Desc, fwActalgo, HIPDNN_PROPAGATE_NAN, 0));

	hipdnnFilterDescriptor_t fc3_Filter_Desc;
	checkCUDNN(hipdnnCreateFilterDescriptor(&fc3_Filter_Desc));
	checkCUDNN(hipdnnSetFilter4dDescriptor(fc3_Filter_Desc, HIPDNN_DATA_FLOAT, HIPDNN_TENSOR_NCHW, lastLayer_numOut, fc2FilterCnt, 1, 1));

	hipdnnConvolutionDescriptor_t fc3_Desc;
	checkCUDNN(hipdnnCreateConvolutionDescriptor(&fc3_Desc));
	checkCUDNN(hipdnnSetConvolution2dDescriptor(fc3_Desc, fcPad_h, fcPad_w, fcStr_h, fcStr_w, fcDil_h, fcDil_w, HIPDNN_CROSS_CORRELATION, HIPDNN_DATA_FLOAT));

	hipdnnTensorDescriptor_t fc3_Tensor;
	checkCUDNN(hipdnnCreateTensorDescriptor(&fc3_Tensor));
	int fwd3Out_n, fwd3Out_c, fwd3Out_h, fwd3Out_w;
	checkCUDNN(hipdnnGetConvolution2dForwardOutputDim(fc3_Desc, fc2_Tensor, fc3_Filter_Desc, &fwd3Out_n, &fwd3Out_c, &fwd3Out_h, &fwd3Out_w));
	checkCUDNN(hipdnnSetTensor4dDescriptor(fc3_Tensor, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, fwd3Out_n, fwd3Out_c, fwd3Out_h, fwd3Out_w));

	hipdnnConvolutionFwdAlgo_t fc3_fwAlgo;
	checkCUDNN(hipdnnGetConvolutionForwardAlgorithm(cudnn, fc2_Tensor, fc3_Filter_Desc, fc3_Desc, fc3_Tensor, HIPDNN_CONVOLUTION_FWD_PREFER_FASTEST, 0, &fc3_fwAlgo));
	size_t fc3_worksSize = 0;
	checkCUDNN(hipdnnGetConvolutionForwardWorkspaceSize(cudnn, fc2_Tensor, fc3_Filter_Desc, fc3_Desc, fc3_Tensor, fc3_fwAlgo, &fc3_worksSize));
	size_t * dev_fc3_works;
	checkCudaErrors(hipMalloc((void**)&dev_fc3_works, fc3_worksSize));

	hipdnnTensorDescriptor_t fc3_biasTensor;
	checkCUDNN(hipdnnCreateTensorDescriptor(&fc3_biasTensor));
	checkCUDNN(hipdnnSetTensor4dDescriptor(fc3_biasTensor, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, 1, lastLayer_numOut, 1, 1));


	/**********BACKPWARD**********/

	float* dev_bw_fc3Bias;
	checkCudaErrors(hipMalloc((void**)&dev_bw_fc3Bias, sizeof(float) * 1 * lastLayer_numOut * 1 * 1));

	float* dev_bwf_fc3_Output;
	checkCudaErrors(hipMalloc((void**)&dev_bwf_fc3_Output, sizeof(float) * lastLayer_numOut * fc2FilterCnt * 1 * 1));

	float* dev_bwd_fc3_Output;
	checkCudaErrors(hipMalloc((void**)&dev_bwd_fc3_Output, sizeof(float) * batchSize * fc2FilterCnt * 1 * 1));

	float* dev_bw_fc2_ActDelta;
	checkCudaErrors(hipMalloc((void**)&dev_bw_fc2_ActDelta, sizeof(float) * batchSize * fc2FilterCnt * 1 * 1));

	float* dev_bw_fc2Bias;
	checkCudaErrors(hipMalloc((void**)&dev_bw_fc2Bias, sizeof(float) * 1 * fc2FilterCnt * 1 * 1));

	float* dev_bwf_fc2_Output;
	checkCudaErrors(hipMalloc((void**)&dev_bwf_fc2_Output, sizeof(float) * fc2FilterCnt * fc1FilterCnt * 1 * 1));

	float* dev_bwd_fc2_Output;
	checkCudaErrors(hipMalloc((void**)&dev_bwd_fc2_Output, sizeof(float) * batchSize * fc1FilterCnt * 1 * 1));

	float* dev_bw_fc1_ActDelta;
	checkCudaErrors(hipMalloc((void**)&dev_bw_fc1_ActDelta, sizeof(float) * batchSize * fc1FilterCnt * 1 * 1));

	float* dev_bw_fc1Bias;
	checkCudaErrors(hipMalloc((void**)&dev_bw_fc1Bias, sizeof(float) * 1 * fc1FilterCnt * 1 * 1));

	float* dev_bwf_fc1_Output;
	checkCudaErrors(hipMalloc((void**)&dev_bwf_fc1_Output, sizeof(float) * fc1FilterCnt * conv2FilterCnt * pool2OutHeight * pool2OutWidth));

	float* dev_bwd_fc1_Output;
	checkCudaErrors(hipMalloc((void**)&dev_bwd_fc1_Output, sizeof(float) * batchSize *  conv2FilterCnt * pool2OutHeight * pool2OutWidth));

	float* dev_bw_conv2pool_ActDelta;
	checkCudaErrors(hipMalloc((void**)&dev_bw_conv2pool_ActDelta, sizeof(float) * batchSize * conv2FilterCnt * pool2OutHeight * pool2OutWidth));

	float* dev_pool2_Delta;
	checkCudaErrors(hipMalloc((void**)&dev_pool2_Delta, sizeof(float) * batchSize * conv2FilterCnt * conv2OutHeight * conv2OutWidth));

	float* dev_bw_conv2_ActDelta;
	checkCudaErrors(hipMalloc((void**)&dev_bw_conv2_ActDelta, sizeof(float) * batchSize * conv2FilterCnt * conv2OutHeight * conv2OutWidth));

	float* dev_bw_conv2Bias;
	checkCudaErrors(hipMalloc((void**)&dev_bw_conv2Bias, sizeof(float) * 1 * conv2FilterCnt * 1 * 1));

	float* dev_bwf_conv2_Output;
	checkCudaErrors(hipMalloc((void**)&dev_bwf_conv2_Output, sizeof(float) * conv2FilterCnt * conv1FilterCnt * conv2FilterHeight * conv2FilterWidth));

	float* dev_bwd_conv2_Output;
	checkCudaErrors(hipMalloc((void**)&dev_bwd_conv2_Output, sizeof(float) * batchSize * conv1FilterCnt * pool1OutHeight * pool1OutWidth));

	float* dev_bw_conv1pool_ActDelta;
	checkCudaErrors(hipMalloc((void**)&dev_bw_conv1pool_ActDelta, sizeof(float) * batchSize * conv1FilterCnt * pool1OutHeight * pool1OutWidth));

	float* dev_pool1_Delta;
	checkCudaErrors(hipMalloc((void**)&dev_pool1_Delta, sizeof(float) * batchSize * conv1FilterCnt * conv1OutHeight * conv1OutWidth));

	float* dev_bw_conv1_ActDelta;
	checkCudaErrors(hipMalloc((void**)&dev_bw_conv1_ActDelta, sizeof(float) * batchSize * conv1FilterCnt * conv1OutHeight * conv1OutWidth));

	float* dev_bw_conv1Bias;
	checkCudaErrors(hipMalloc((void**)&dev_bw_conv1Bias, sizeof(float) * 1 * conv1FilterCnt * 1 * 1));

	float* dev_bwf_conv1_Output;
	checkCudaErrors(hipMalloc((void**)&dev_bwf_conv1_Output, sizeof(float) * conv1FilterCnt * input_channelCnt * conv1FilterHeight * conv1FilterWidth));


	//weight update momentum
	float* dev_fc3FiltMentum;
	checkCudaErrors(hipMalloc((void**)&dev_fc3FiltMentum, sizeof(float) * lastLayer_numOut * fc2FilterCnt * 1 * 1));
	MomentumInitialize << <(lastLayer_numOut * fc2FilterCnt * 1 * 1 + BW - 1) / BW, BW >> > (lastLayer_numOut, fc2FilterCnt, 1, 1, dev_fc3FiltMentum);

	float* dev_fc3BiasMentum;
	checkCudaErrors(hipMalloc((void**)&dev_fc3BiasMentum, sizeof(float) * 1 * lastLayer_numOut * 1 * 1));
	MomentumInitialize << <(1 * lastLayer_numOut * 1 * 1 + BW - 1) / BW, BW >> > (1, lastLayer_numOut, 1, 1, dev_fc3BiasMentum);

	float* dev_fc2FiltMentum;
	checkCudaErrors(hipMalloc((void**)&dev_fc2FiltMentum, sizeof(float) * fc2FilterCnt * fc1FilterCnt * 1 * 1));
	MomentumInitialize << <(fc2FilterCnt * fc1FilterCnt * 1 * 1 + BW - 1) / BW, BW >> > (fc2FilterCnt, fc1FilterCnt, 1, 1, dev_fc2FiltMentum);

	float* dev_fc2BiasMentum;
	checkCudaErrors(hipMalloc((void**)&dev_fc2BiasMentum, sizeof(float) * 1 * fc2FilterCnt * 1 * 1));
	MomentumInitialize << <(1 * fc2FilterCnt * 1 * 1 + BW - 1) / BW, BW >> > (1, fc2FilterCnt, 1, 1, dev_fc2BiasMentum);

	float* dev_fc1FiltMentum;
	checkCudaErrors(hipMalloc((void**)&dev_fc1FiltMentum, sizeof(float) * fc1FilterCnt * conv2FilterCnt * pool2OutHeight * pool2OutWidth));
	MomentumInitialize << <(fc1FilterCnt * conv2FilterCnt * pool2OutHeight * pool2OutWidth + BW - 1) / BW, BW >> > (fc1FilterCnt, conv2FilterCnt, pool2OutHeight, pool2OutWidth, dev_fc1FiltMentum);

	float* dev_fc1BiasMentum;
	checkCudaErrors(hipMalloc((void**)&dev_fc1BiasMentum, sizeof(float) * 1 * fc1FilterCnt * 1 * 1));
	MomentumInitialize << <(1 * fc1FilterCnt * 1 * 1 + BW - 1) / BW, BW >> > (1, fc1FilterCnt, 1, 1, dev_fc1BiasMentum);

	float* dev_conv2FiltMentum;
	checkCudaErrors(hipMalloc((void**)&dev_conv2FiltMentum, sizeof(float) * conv2FilterCnt * conv1FilterCnt * conv2FilterHeight * conv2FilterWidth));
	MomentumInitialize << <(conv2FilterCnt * conv1FilterCnt * conv2FilterHeight * conv2FilterWidth + BW - 1) / BW, BW >> > (conv2FilterCnt, conv1FilterCnt, conv2FilterHeight, conv2FilterWidth, dev_conv2FiltMentum);

	float* dev_conv2BiasMentum;
	checkCudaErrors(hipMalloc((void**)&dev_conv2BiasMentum, sizeof(float) * 1 * conv2FilterCnt * 1 * 1));
	MomentumInitialize << <(1 * conv2FilterCnt * 1 * 1 + BW - 1) / BW, BW >> > (1, conv2FilterCnt, 1, 1, dev_conv2BiasMentum);

	float* dev_conv1FiltMentum;
	checkCudaErrors(hipMalloc((void**)&dev_conv1FiltMentum, sizeof(float) * conv1FilterCnt * input_channelCnt * conv1FilterHeight * conv1FilterWidth));
	MomentumInitialize << <(conv1FilterCnt * input_channelCnt * conv1FilterHeight * conv1FilterWidth + BW - 1) / BW, BW >> > (conv1FilterCnt, input_channelCnt, conv1FilterHeight, conv1FilterWidth, dev_conv1FiltMentum);

	float* dev_conv1BiasMentum;
	checkCudaErrors(hipMalloc((void**)&dev_conv1BiasMentum, sizeof(float) * 1 * conv1FilterCnt * 1 * 1));
	MomentumInitialize << <(1 * conv1FilterCnt * 1 * 1 + BW - 1) / BW, BW >> > (1, conv1FilterCnt, 1, 1, dev_conv1BiasMentum);

	//=======================


	hipdnnConvolutionBwdFilterAlgo_t fc3_bwFAlgo;
	checkCUDNN(hipdnnGetConvolutionBackwardFilterAlgorithm(cudnn, fc2_Tensor, fc3_Tensor, fc3_Desc, fc3_Filter_Desc, HIPDNN_CONVOLUTION_BWD_FILTER_PREFER_FASTEST, 0, &fc3_bwFAlgo));
	size_t fc3_bwFworksSize = 0;
	checkCUDNN(hipdnnGetConvolutionBackwardFilterWorkspaceSize(cudnn, fc2_Tensor, fc3_Tensor, fc3_Desc, fc3_Filter_Desc, fc3_bwFAlgo, &fc3_bwFworksSize));
	size_t* dev_fc3_bwFworks;
	checkCudaErrors(hipMalloc((void**)&dev_fc3_bwFworks, fc3_bwFworksSize));

	hipdnnConvolutionBwdDataAlgo_t fc3_bwDAlgo;
	checkCUDNN(hipdnnGetConvolutionBackwardDataAlgorithm(cudnn, fc3_Filter_Desc, fc3_Tensor, fc3_Desc, fc2_Tensor, HIPDNN_CONVOLUTION_BWD_DATA_PREFER_FASTEST, 0, &fc3_bwDAlgo));
	size_t fc3_bwDworksSize = 0;
	checkCUDNN(hipdnnGetConvolutionBackwardDataWorkspaceSize(cudnn, fc3_Filter_Desc, fc3_Tensor, fc3_Desc, fc2_Tensor, fc3_bwDAlgo, &fc3_bwDworksSize));
	size_t* dev_fc3_bwDworks;
	checkCudaErrors(hipMalloc((void**)&dev_fc3_bwDworks, fc3_bwDworksSize));

	hipdnnConvolutionBwdFilterAlgo_t fc2_bwFAlgo;
	checkCUDNN(hipdnnGetConvolutionBackwardFilterAlgorithm(cudnn, fc1_Tensor, fc2_Tensor, fc2_Desc, fc2_Filter_Desc, HIPDNN_CONVOLUTION_BWD_FILTER_PREFER_FASTEST, 0, &fc2_bwFAlgo));
	size_t fc2_bwFworksSize = 0;
	checkCUDNN(hipdnnGetConvolutionBackwardFilterWorkspaceSize(cudnn, fc1_Tensor, fc2_Tensor, fc2_Desc, fc2_Filter_Desc, fc2_bwFAlgo, &fc2_bwFworksSize));
	size_t* dev_fc2_bwFworks;
	checkCudaErrors(hipMalloc((void**)&dev_fc2_bwFworks, fc2_bwFworksSize));

	hipdnnConvolutionBwdDataAlgo_t fc2_bwDAlgo;
	checkCUDNN(hipdnnGetConvolutionBackwardDataAlgorithm(cudnn, fc2_Filter_Desc, fc2_Tensor, fc2_Desc, fc1_Tensor, HIPDNN_CONVOLUTION_BWD_DATA_PREFER_FASTEST, 0, &fc2_bwDAlgo));
	size_t fc2_bwDworksSize = 0;
	checkCUDNN(hipdnnGetConvolutionBackwardDataWorkspaceSize(cudnn, fc2_Filter_Desc, fc2_Tensor, fc2_Desc, fc1_Tensor, fc2_bwDAlgo, &fc2_bwDworksSize));
	size_t* dev_fc2_bwDworks;
	checkCudaErrors(hipMalloc((void**)&dev_fc2_bwDworks, fc2_bwDworksSize));

	hipdnnConvolutionBwdFilterAlgo_t fc1_bwFAlgo;
	checkCUDNN(hipdnnGetConvolutionBackwardFilterAlgorithm(cudnn, pool2_Tensor, fc1_Tensor, fc1_Desc, fc1_Filter_Desc, HIPDNN_CONVOLUTION_BWD_FILTER_PREFER_FASTEST, 0, &fc1_bwFAlgo));
	size_t fc1_bwFworksSize = 0;
	checkCUDNN(hipdnnGetConvolutionBackwardFilterWorkspaceSize(cudnn, pool2_Tensor, fc1_Tensor, fc1_Desc, fc1_Filter_Desc, fc1_bwFAlgo, &fc1_bwFworksSize));
	size_t* dev_fc1_bwFworks;
	checkCudaErrors(hipMalloc((void**)&dev_fc1_bwFworks, fc1_bwFworksSize));

	hipdnnConvolutionBwdDataAlgo_t fc1_bwDAlgo;
	checkCUDNN(hipdnnGetConvolutionBackwardDataAlgorithm(cudnn, fc1_Filter_Desc, fc1_Tensor, fc1_Desc, pool2_Tensor, HIPDNN_CONVOLUTION_BWD_DATA_PREFER_FASTEST, 0, &fc1_bwDAlgo));
	size_t fc1_bwDworksSize = 0;
	checkCUDNN(hipdnnGetConvolutionBackwardDataWorkspaceSize(cudnn, fc1_Filter_Desc, fc1_Tensor, fc1_Desc, pool2_Tensor, fc1_bwDAlgo, &fc1_bwDworksSize));
	size_t* dev_fc1_bwDworks;
	checkCudaErrors(hipMalloc((void**)&dev_fc1_bwDworks, fc1_bwDworksSize));

	hipdnnConvolutionBwdFilterAlgo_t conv2_bwFAlgo;
	checkCUDNN(hipdnnGetConvolutionBackwardFilterAlgorithm(cudnn, pool1_Tensor, conv2_Tensor, conv2_Desc, conv2_Filter_Desc, HIPDNN_CONVOLUTION_BWD_FILTER_PREFER_FASTEST, 0, &conv2_bwFAlgo));
	size_t conv2_bwFworksSize = 0;
	checkCUDNN(hipdnnGetConvolutionBackwardFilterWorkspaceSize(cudnn, pool1_Tensor, conv2_Tensor, conv2_Desc, conv2_Filter_Desc, conv2_bwFAlgo, &conv2_bwFworksSize));
	size_t * dev_conv2_bwFworks;
	checkCudaErrors(hipMalloc((void**)&dev_conv2_bwFworks, conv2_bwFworksSize));

	hipdnnConvolutionBwdDataAlgo_t conv2_bwDAlgo;
	checkCUDNN(hipdnnGetConvolutionBackwardDataAlgorithm(cudnn, conv2_Filter_Desc, conv2_Tensor, conv2_Desc, pool1_Tensor, HIPDNN_CONVOLUTION_BWD_DATA_PREFER_FASTEST, 0, &conv2_bwDAlgo));
	size_t conv2_bwDworksSize = 0;
	checkCUDNN(hipdnnGetConvolutionBackwardDataWorkspaceSize(cudnn, conv2_Filter_Desc, conv2_Tensor, conv2_Desc, pool1_Tensor, conv2_bwDAlgo, &conv2_bwDworksSize));
	size_t* dev_conv2_bwDworks;
	checkCudaErrors(hipMalloc((void**)&dev_conv2_bwDworks, conv2_bwDworksSize));

	hipdnnConvolutionBwdFilterAlgo_t conv1_bwFAlgo;
	checkCUDNN(hipdnnGetConvolutionBackwardFilterAlgorithm(cudnn, input_Tensor, conv1_Tensor, conv1_Desc, conv1_Filter_Desc, HIPDNN_CONVOLUTION_BWD_FILTER_PREFER_FASTEST, 0, &conv1_bwFAlgo));
	size_t conv1_bwFworksSize = 0;
	checkCUDNN(hipdnnGetConvolutionBackwardFilterWorkspaceSize(cudnn, input_Tensor, conv1_Tensor, conv1_Desc, conv1_Filter_Desc, conv1_bwFAlgo, &conv1_bwFworksSize));
	size_t * dev_conv1_bwFworks;
	checkCudaErrors(hipMalloc((void**)&dev_conv1_bwFworks, conv1_bwFworksSize));




	float * dev_dloss;


	float* dev_target;
	hipMalloc((void**)&dev_target, sizeof(float) * batchSize);

	float* dev_input;
	hipMalloc((void**)&dev_input, sizeof(float) * batchSize * input_channelCnt * imageHeight * imageWidth);

	cout << "======================  �н� ����  ==========================" << endl;

	startTime_train = clock();

	for (int iter = 0; iter < epoch; iter++)
	{


		float* yhat = new float[batchSize* num_labels];// soft max ��� �� 
		int* predicted = new int[ImageNum]; // ����Ʈ �ƽ� ��� ������ ���� ū ���� ��� �ִ� ��ġ(�ε���)�� �����ϴ� �迭
		int count = 0;


		for (int a = 0; a < ImageNum / batchSize; a++) { // ��ġ ��� ���� 

			for (int i = 0; i < batchSize; i++) {
				unsigned char* temp_train = ImgBox[i + (batchSize * a)].first.data;
				for (int c = 0; c < input_channelCnt; c++) {
					for (int y = 0; y < imageHeight; y++) {
						for (int x = 0; x < imageWidth; x++) {
							Input_train[i * input_channelCnt * imageHeight * imageWidth + c * imageHeight * imageWidth + y * imageWidth + x] = temp_train[input_channelCnt * imageHeight * x + input_channelCnt * y + c] / 255.0;
						}
					}
				}

			}

			for (int i = 0; i < batchSize; i++)
			{
				target_train_batch[i] = target_train[i + (batchSize * a)];
			}



			hipMemcpy(dev_target, target_train_batch, sizeof(float) * batchSize, hipMemcpyHostToDevice);
			hipMemcpy(dev_input, Input_train, sizeof(float) * batchSize * input_channelCnt * imageHeight * imageWidth, hipMemcpyHostToDevice);

			//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
			//GPU train ��� �� �ڸ� 
			hipdnnConvolutionForward(cudnn, &alpha, input_Tensor, dev_input, conv1_Filter_Desc, dev_conv1_Filter, conv1_Desc, conv1_fwAlgo, dev_conv1_works, conv1_worksSize, &beta, conv1_Tensor, dev_conv1_Output);
			hipdnnAddTensor(cudnn, &alpha, conv1_biasTensor, dev_conv1_Bias, &alpha, conv1_Tensor, dev_conv1_Output);
			hipdnnActivationForward(cudnn, conv1_Act_Desc, &alpha, conv1_Tensor, dev_conv1_Output, &beta, conv1_Tensor, dev_conv1Act_output);
			hipdnnPoolingForward(cudnn, pool1_Desc, &alpha, conv1_Tensor, dev_conv1Act_output, &beta, pool1_Tensor, dev_pool1_Output);
			hipdnnActivationForward(cudnn, conv1pool_Act_Desc, &alpha, pool1_Tensor, dev_pool1_Output, &beta, pool1_Tensor, dev_conv1pool_Act_output);
			//conv2 forward
			hipdnnConvolutionForward(cudnn, &alpha, pool1_Tensor, dev_conv1pool_Act_output, conv2_Filter_Desc, dev_conv2_Filter, conv2_Desc, conv2_fwAlgo, dev_conv2_works, conv2_worksSize, &beta, conv2_Tensor, dev_conv2_Output);
			hipdnnAddTensor(cudnn, &alpha, conv2_biasTensor, dev_conv2_Bias, &alpha, conv2_Tensor, dev_conv2_Output);
			hipdnnActivationForward(cudnn, conv2_Act_Desc, &alpha, conv2_Tensor, dev_conv2_Output, &beta, conv2_Tensor, dev_conv2Act_output);
			hipdnnPoolingForward(cudnn, pool2_Desc, &alpha, conv2_Tensor, dev_conv2Act_output, &beta, pool2_Tensor, dev_pool2_Output);
			hipdnnActivationForward(cudnn, conv2pool_Act_Desc, &alpha, pool2_Tensor, dev_pool2_Output, &beta, pool2_Tensor, dev_conv2pool_Act_output);






			//fc1 forward
			hipdnnConvolutionForward(cudnn, &alpha, pool2_Tensor, dev_conv2pool_Act_output, fc1_Filter_Desc, dev_fc1_Filter, fc1_Desc, fc1_fwAlgo, dev_fc1_works, fc1_worksSize, &beta, fc1_Tensor, dev_fc1_Output);
			hipdnnAddTensor(cudnn, &alpha, fc1_biasTensor, dev_fc1_Bias, &alpha, fc1_Tensor, dev_fc1_Output);
			hipdnnActivationForward(cudnn, fc1_Act_Desc, &alpha, fc1_Tensor, dev_fc1_Output, &beta, fc1_Tensor, dev_fc1_Actout);






			//fc2 forward
			hipdnnConvolutionForward(cudnn, &alpha, fc1_Tensor, dev_fc1_Actout, fc2_Filter_Desc, dev_fc2_Filter, fc2_Desc, fc2_fwAlgo, dev_fc2_works, fc2_worksSize, &beta, fc2_Tensor, dev_fc2_Output);
			hipdnnAddTensor(cudnn, &alpha, fc2_biasTensor, dev_fc2_Bias, &alpha, fc2_Tensor, dev_fc2_Output);
			hipdnnActivationForward(cudnn, fc2_Act_Desc, &alpha, fc2_Tensor, dev_fc2_Output, &beta, fc2_Tensor, dev_fc2_Actout);





			//fc3 forward
			hipdnnConvolutionForward(cudnn, &alpha, fc2_Tensor, dev_fc2_Actout, fc3_Filter_Desc, dev_fc3_Filter, fc3_Desc, fc3_fwAlgo, dev_fc3_works, fc3_worksSize, &beta, fc3_Tensor, dev_fc3_Output);
			hipdnnAddTensor(cudnn, &alpha, fc3_biasTensor, dev_fc3_Bias, &alpha, fc3_Tensor, dev_fc3_Output);
			hipdnnSoftmaxForward(cudnn, sftAlgo, sftMode, &alpha, fc3_Tensor, dev_fc3_Output, &beta, fc3_Tensor, dev_smaxOutput);
			dev_dloss = dev_smaxOutput;





			hipMemcpy(yhat, dev_smaxOutput, sizeof(float) * batchSize * num_labels, hipMemcpyDeviceToHost);

			//one hot ��ġ ã��


			for (size_t i = 0; i < batchSize; i++) {
				float temp = yhat[i * 10];// ������ ù��° ���� �ӽ� ������ ����
				int indexJ = 0; // ���� ū���� ã�� ���� ��� �Ǵ� ��ġ �ε���

				for (size_t j = 0; j < num_labels - 1; j++) {
					if (temp > yhat[i * 10 + j + 1]) // �ӽ� ������ �־��� ���� ��
					{
						yhat[i * 10 + j + 1] = 0; // �ӽ� ������ ��� �ִ� ������ �۴ٸ� 0 �Է�
					}
					else                      // �ӽ� ������ ��� �ִ� ������ ũ�ٸ�
					{
						temp = yhat[i * 10 + j + 1]; // �ӽ� ������ �ش� ���� ����
						yhat[i * 10 + indexJ] = 0; // �ӽ� ������ ������ ��� �ִ� ���� �ε����� �̿��Ͽ� ���� �� ��ġ�� 0 �Է�
						indexJ = j + 1; // ���� ū ��(���� ���� ��) ��ġ�� �ε��� ������ ����
					}
				}

				predicted[i] = indexJ; // �ش� �̹����� ����Ʈ �ƽ� ���� ���� ū ���� ���� ��ġ �ε����� �迭�� ����
			}








			// Ŀ�� �Լ� ( ����(=dloss=dy)�� ���) 
			SoftmaxLossBackprop << <(batchSize + BW - 1) / BW, BW >> > (dev_target, num_labels, batchSize, dev_dloss);


			//fc3 back
			hipdnnConvolutionBackwardBias(cudnn, &alpha, fc3_Tensor, dev_dloss, &beta, fc3_biasTensor, dev_bw_fc3Bias);
			hipdnnConvolutionBackwardFilter(cudnn, &alpha, fc2_Tensor, dev_fc2_Actout, fc3_Tensor, dev_dloss, fc3_Desc,
				fc3_bwFAlgo, dev_fc3_bwFworks, fc3_bwFworksSize, &beta, fc3_Filter_Desc, dev_bwf_fc3_Output);
			hipdnnConvolutionBackwardData(cudnn, &alpha, fc3_Filter_Desc, dev_fc3_Filter, fc3_Tensor, dev_dloss,
				fc3_Desc, fc3_bwDAlgo, dev_fc3_bwDworks, fc3_bwDworksSize, &beta, fc2_Tensor, dev_bwd_fc3_Output);




			//fc2 back
			hipdnnActivationBackward(cudnn, fc2_Act_Desc, &alpha, fc2_Tensor, dev_fc2_Actout, fc2_Tensor,
				dev_bwd_fc3_Output, fc2_Tensor, dev_fc2_Output, &beta, fc2_Tensor, dev_bw_fc2_ActDelta);
			hipdnnConvolutionBackwardBias(cudnn, &alpha, fc2_Tensor, dev_bw_fc2_ActDelta, &beta, fc2_biasTensor, dev_bw_fc2Bias);
			hipdnnConvolutionBackwardFilter(cudnn, &alpha, fc1_Tensor, dev_fc1_Actout, fc2_Tensor, dev_bw_fc2_ActDelta, fc2_Desc,
				fc2_bwFAlgo, dev_fc2_bwFworks, fc2_bwFworksSize, &beta, fc2_Filter_Desc, dev_bwf_fc2_Output);
			hipdnnConvolutionBackwardData(cudnn, &alpha, fc2_Filter_Desc, dev_fc2_Filter, fc2_Tensor, dev_bw_fc2_ActDelta,
				fc2_Desc, fc2_bwDAlgo, dev_fc2_bwDworks, fc2_bwDworksSize, &beta, fc1_Tensor, dev_bwd_fc2_Output);




			//fc1 back
			hipdnnActivationBackward(cudnn, fc1_Act_Desc, &alpha, fc1_Tensor, dev_fc1_Actout, fc1_Tensor,
				dev_bwd_fc2_Output, fc1_Tensor, dev_fc1_Output, &beta, fc1_Tensor, dev_bw_fc1_ActDelta);
			hipdnnConvolutionBackwardBias(cudnn, &alpha, fc1_Tensor, dev_bw_fc1_ActDelta, &beta, fc1_biasTensor, dev_bw_fc1Bias);
			hipdnnConvolutionBackwardFilter(cudnn, &alpha, pool2_Tensor, dev_conv2pool_Act_output, fc1_Tensor, dev_bw_fc1_ActDelta, fc1_Desc,
				fc1_bwFAlgo, dev_fc1_bwFworks, fc1_bwFworksSize, &beta, fc1_Filter_Desc, dev_bwf_fc1_Output);
			hipdnnConvolutionBackwardData(cudnn, &alpha, fc1_Filter_Desc, dev_fc1_Filter, fc1_Tensor, dev_bw_fc1_ActDelta,
				fc1_Desc, fc1_bwDAlgo, dev_fc1_bwDworks, fc1_bwDworksSize, &beta, pool2_Tensor, dev_bwd_fc1_Output);


			//conv2 back
			hipdnnActivationBackward(cudnn, conv2pool_Act_Desc, &alpha, pool2_Tensor, dev_conv2pool_Act_output, pool2_Tensor,
				dev_bwd_fc1_Output, pool2_Tensor, dev_pool2_Output, &beta, pool2_Tensor, dev_bw_conv2pool_ActDelta);
			hipdnnPoolingBackward(cudnn, pool2_Desc, &alpha, pool2_Tensor, dev_pool2_Output, pool2_Tensor,
				dev_bw_conv2pool_ActDelta, conv2_Tensor, dev_conv2_Output, &beta, conv2_Tensor, dev_pool2_Delta);
			hipdnnActivationBackward(cudnn, conv2_Act_Desc, &alpha, conv2_Tensor, dev_conv2Act_output, conv2_Tensor,
				dev_pool2_Delta, conv2_Tensor, dev_conv2_Output, &beta, conv2_Tensor, dev_bw_conv2_ActDelta);
			hipdnnConvolutionBackwardBias(cudnn, &alpha, conv2_Tensor, dev_bw_conv2_ActDelta, &beta, conv2_biasTensor, dev_bw_conv2Bias);
			hipdnnConvolutionBackwardFilter(cudnn, &alpha, pool1_Tensor, dev_conv1pool_Act_output, conv2_Tensor,
				dev_bw_conv2_ActDelta, conv2_Desc, conv2_bwFAlgo, dev_conv2_bwFworks,
				conv2_bwFworksSize, &beta, conv2_Filter_Desc, dev_bwf_conv2_Output);
			hipdnnConvolutionBackwardData(cudnn, &alpha, conv2_Filter_Desc, dev_conv2_Filter, conv2_Tensor, dev_bw_conv2_ActDelta,
				conv2_Desc, conv2_bwDAlgo, dev_conv2_bwDworks, conv2_bwDworksSize, &beta, pool1_Tensor, dev_bwd_conv2_Output);

			//conv1 back
			hipdnnActivationBackward(cudnn, conv1pool_Act_Desc, &alpha, pool1_Tensor, dev_conv1pool_Act_output, pool1_Tensor,
				dev_bwd_conv2_Output, pool1_Tensor, dev_pool1_Output, &beta, pool1_Tensor, dev_bw_conv1pool_ActDelta);
			hipdnnPoolingBackward(cudnn, pool1_Desc, &alpha, pool1_Tensor, dev_pool1_Output, pool1_Tensor,
				dev_bw_conv1pool_ActDelta, conv1_Tensor, dev_conv1_Output, &beta, conv1_Tensor, dev_pool1_Delta);
			hipdnnActivationBackward(cudnn, conv1_Act_Desc, &alpha, conv1_Tensor, dev_conv1Act_output, conv1_Tensor,
				dev_pool1_Delta, conv1_Tensor, dev_conv1_Output, &beta, conv1_Tensor, dev_bw_conv1_ActDelta);
			hipdnnConvolutionBackwardBias(cudnn, &alpha, conv1_Tensor, dev_bw_conv1_ActDelta, &beta, conv1_biasTensor, dev_bw_conv1Bias);
			hipdnnConvolutionBackwardFilter(cudnn, &alpha, input_Tensor, dev_input, conv1_Tensor,
				dev_bw_conv1_ActDelta, conv1_Desc, conv1_bwFAlgo, dev_conv1_bwFworks,
				conv1_bwFworksSize, &beta, conv1_Filter_Desc, dev_bwf_conv1_Output);



			//learning_rate = static_cast<float>(learning_rate * pow((1.0 + 0.0001*iter), 0.75));


			// Fully connected 3
			hipblasSscal(cublasHandle, static_cast<int>(lastLayer_numOut * fc2FilterCnt * 1 * 1), &momentum, dev_fc3FiltMentum, 1);
			hipblasSaxpy(cublasHandle, static_cast<int>(lastLayer_numOut * fc2FilterCnt * 1 * 1), &learning_rate, dev_bwf_fc3_Output, 1, dev_fc3FiltMentum, 1);
			hipblasSaxpy(cublasHandle, static_cast<int>(lastLayer_numOut * fc2FilterCnt * 1 * 1), &eta, dev_fc3FiltMentum, 1, dev_fc3_Filter, 1);

			hipblasSscal(cublasHandle, static_cast<int>(1 * lastLayer_numOut * 1 * 1), &momentum, dev_fc3BiasMentum, 1);
			hipblasSaxpy(cublasHandle, static_cast<int>(1 * lastLayer_numOut * 1 * 1), &learning_rate, dev_bw_fc3Bias, 1, dev_fc3BiasMentum, 1);
			hipblasSaxpy(cublasHandle, static_cast<int>(1 * lastLayer_numOut * 1 * 1), &eta, dev_fc3BiasMentum, 1, dev_fc3_Bias, 1);

			// Fully connected 2
			hipblasSscal(cublasHandle, static_cast<int>(fc2FilterCnt * fc1FilterCnt * 1 * 1), &momentum, dev_fc2FiltMentum, 1);
			hipblasSaxpy(cublasHandle, static_cast<int>(fc2FilterCnt * fc1FilterCnt * 1 * 1), &learning_rate, dev_bwf_fc2_Output, 1, dev_fc2FiltMentum, 1);
			hipblasSaxpy(cublasHandle, static_cast<int>(fc2FilterCnt * fc1FilterCnt * 1 * 1), &eta, dev_fc2FiltMentum, 1, dev_fc2_Filter, 1);

			hipblasSscal(cublasHandle, static_cast<int>(1 * fc2FilterCnt * 1 * 1), &momentum, dev_fc2BiasMentum, 1);
			hipblasSaxpy(cublasHandle, static_cast<int>(1 * fc2FilterCnt * 1 * 1), &learning_rate, dev_bw_fc2Bias, 1, dev_fc2BiasMentum, 1);
			hipblasSaxpy(cublasHandle, static_cast<int>(1 * fc2FilterCnt * 1 * 1), &eta, dev_fc2BiasMentum, 1, dev_fc2_Bias, 1);

			// Fully connected 1
			hipblasSscal(cublasHandle, static_cast<int>(fc1FilterCnt * conv2FilterCnt * pool2OutHeight * pool2OutWidth), &momentum, dev_fc1FiltMentum, 1);
			hipblasSaxpy(cublasHandle, static_cast<int>(fc1FilterCnt * conv2FilterCnt * pool2OutHeight * pool2OutWidth), &learning_rate, dev_bwf_fc1_Output, 1, dev_fc1FiltMentum, 1);
			hipblasSaxpy(cublasHandle, static_cast<int>(fc1FilterCnt * conv2FilterCnt * pool2OutHeight * pool2OutWidth), &eta, dev_fc1FiltMentum, 1, dev_fc1_Filter, 1);

			hipblasSscal(cublasHandle, static_cast<int>(1 * fc1FilterCnt * 1 * 1), &momentum, dev_fc1BiasMentum, 1);
			hipblasSaxpy(cublasHandle, static_cast<int>(1 * fc1FilterCnt * 1 * 1), &learning_rate, dev_bw_fc1Bias, 1, dev_fc1BiasMentum, 1);
			hipblasSaxpy(cublasHandle, static_cast<int>(1 * fc1FilterCnt * 1 * 1), &eta, dev_fc1BiasMentum, 1, dev_fc1_Bias, 1);

			// Conv2
			hipblasSscal(cublasHandle, static_cast<int>(conv2FilterCnt * conv1FilterCnt * conv2FilterHeight * conv2FilterWidth), &momentum, dev_conv2FiltMentum, 1);
			hipblasSaxpy(cublasHandle, static_cast<int>(conv2FilterCnt * conv1FilterCnt * conv2FilterHeight * conv2FilterWidth), &learning_rate, dev_bwf_conv2_Output, 1, dev_conv2FiltMentum, 1);
			hipblasSaxpy(cublasHandle, static_cast<int>(conv2FilterCnt * conv1FilterCnt * conv2FilterHeight * conv2FilterWidth), &eta, dev_conv2FiltMentum, 1, dev_conv2_Filter, 1);

			hipblasSscal(cublasHandle, static_cast<int>(1 * conv2FilterCnt * 1 * 1), &momentum, dev_conv2BiasMentum, 1);
			hipblasSaxpy(cublasHandle, static_cast<int>(1 * conv2FilterCnt * 1 * 1), &learning_rate, dev_bw_conv2Bias, 1, dev_conv2BiasMentum, 1);
			hipblasSaxpy(cublasHandle, static_cast<int>(1 * conv2FilterCnt * 1 * 1), &eta, dev_conv2BiasMentum, 1, dev_conv2_Bias, 1);

			// Conv1
			hipblasSscal(cublasHandle, static_cast<int>(conv1FilterCnt * input_channelCnt * conv1FilterHeight * conv1FilterWidth), &momentum, dev_conv1FiltMentum, 1);
			hipblasSaxpy(cublasHandle, static_cast<int>(conv1FilterCnt * input_channelCnt * conv1FilterHeight * conv1FilterWidth), &learning_rate, dev_bwf_conv1_Output, 1, dev_conv1FiltMentum, 1);
			hipblasSaxpy(cublasHandle, static_cast<int>(conv1FilterCnt * input_channelCnt * conv1FilterHeight * conv1FilterWidth), &eta, dev_conv1FiltMentum, 1, dev_conv1_Filter, 1);

			hipblasSscal(cublasHandle, static_cast<int>(1 * conv1FilterCnt * 1 * 1), &momentum, dev_conv1BiasMentum, 1);
			hipblasSaxpy(cublasHandle, static_cast<int>(1 * conv1FilterCnt * 1 * 1), &learning_rate, dev_bw_conv1Bias, 1, dev_conv1BiasMentum, 1);
			hipblasSaxpy(cublasHandle, static_cast<int>(1 * conv1FilterCnt * 1 * 1), &eta, dev_conv1BiasMentum, 1, dev_conv1_Bias, 1);



			//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

			//accuracy ���

			for (size_t i = 0; i < batchSize; i++) {
				if (predicted[i] == target_train_batch[i])
					count++;
			}


		}



		cout << "# EPOCH (" << setw(3) << iter + 1 << " / " << epoch << " ) , " << "Train Set Accuracy ( " << setw(5) << (count*100.0 / ImageNum) << " %)" << " , ���� ���� :: " << setw(5) << count << endl;

	}


	endTime_train = clock();


	///////////////////////////////////////////
	//test ���� ������ �� �غ� 


	vector<string> LabelBox_test; // �� ������ ���� ����
	vector<pair<int, string>> LabelTable_test; // �󺧸� ���� �ѹ� �ο�
	float* target_test = new float[ImageNum_test]; // target �� , �󺧿� ���� ������ �ѹ� ���� ��� �迭


												   // �󺧿� ��ȣ �ο��� ���� LabelBox ���Ϳ� �� ���� �ϰ� ���� �� �ߺ� ����
	for (int i = 0; i < ImageNum_test; i++) {
		LabelBox_test.push_back(ImgBox_test[i].second);
		//std::cout<< "�� ��� :: " << ImgBox[i].second << std::endl; // �Է¹���������� �� ��� -> ���� "�� ��� :: automobile"
	}

	sort(LabelBox_test.begin(), LabelBox_test.end());
	LabelBox_test.erase(unique(LabelBox_test.begin(), LabelBox_test.end()), LabelBox_test.end());
	int nLabelBoxSize_test = LabelBox_test.size();

	// �� ��ȣ �ο�
	for (int i = 0; i < nLabelBoxSize_test; i++) {
		LabelTable_test.push_back({ { i },{ LabelBox_test[i] } });
		//std::cout << "LabelBox :: " << LabelBox[i] << std::endl;// -> ���� "LabelBox :: truck"
	}

	//target ����
	for (int i = 0; i < ImageNum_test; i++) {
		for (int j = 0; j < LabelTable_test.size(); j++) {
			if (ImgBox_test[i].second == LabelTable_test[j].second) {
				target_test[i] = LabelTable_test[j].first;
			}
		}
	}


	//�Էº���
	float* target_test_batch = new float[batch_size_test];

	float* Input_test = new float[batch_size_test * input_channelCnt * imageHeight * imageWidth];

	float* yhat = new float[batch_size_test* num_labels];// soft max ��� �� 

	int* predicted = new int[ImageNum_test]; // ����Ʈ �ƽ� ��� ������ ���� ū ���� ��� �ִ� ��ġ(�ε���)�� �����ϴ� �迭
	int count = 0;

	float* dev_input_test;
	checkCudaErrors(hipMalloc((void**)&dev_input_test, sizeof(float) * batch_size_test * input_channelCnt * imageHeight * imageWidth));

	for (int a = 0; a < ImageNum_test / batch_size_test; a++) { // ��ġ ��� ����


		for (int i = 0; i < batch_size_test; i++) {
			unsigned char* temp_test = ImgBox_test[i + (batch_size_test * a)].first.data;

			for (int c = 0; c < input_channelCnt; c++) {
				for (int y = 0; y < imageHeight; y++) {
					for (int x = 0; x < imageWidth; x++) {
						Input_test[i * input_channelCnt * imageHeight * imageWidth + c * imageHeight * imageWidth + y * imageWidth + x] = temp_test[input_channelCnt * imageHeight * x + input_channelCnt * y + c] / 255.0;
					}
				}
			}
		}
		for (int i = 0; i < batch_size_test; i++) {
			target_test_batch[i] = target_test[i + (batch_size_test * a)];
		}


		checkCudaErrors(hipMemcpy(dev_input_test, Input_test, sizeof(float) * batch_size_test * input_channelCnt * imageHeight * imageWidth, hipMemcpyHostToDevice));
		//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
		//GPU test ��� �� �ڸ�
		//conv1 forward
		hipdnnConvolutionForward(cudnn, &alpha, input_Tensor, dev_input_test, conv1_Filter_Desc, dev_conv1_Filter, conv1_Desc, conv1_fwAlgo, dev_conv1_works, conv1_worksSize, &beta, conv1_Tensor, dev_conv1_Output);
		hipdnnAddTensor(cudnn, &alpha, conv1_biasTensor, dev_conv1_Bias, &alpha, conv1_Tensor, dev_conv1_Output);
		hipdnnActivationForward(cudnn, conv1_Act_Desc, &alpha, conv1_Tensor, dev_conv1_Output, &beta, conv1_Tensor, dev_conv1Act_output);
		hipdnnPoolingForward(cudnn, pool1_Desc, &alpha, conv1_Tensor, dev_conv1Act_output, &beta, pool1_Tensor, dev_pool1_Output);
		hipdnnActivationForward(cudnn, conv1pool_Act_Desc, &alpha, pool1_Tensor, dev_pool1_Output, &beta, pool1_Tensor, dev_conv1pool_Act_output);
		//conv2 forward
		hipdnnConvolutionForward(cudnn, &alpha, pool1_Tensor, dev_conv1pool_Act_output, conv2_Filter_Desc, dev_conv2_Filter, conv2_Desc, conv2_fwAlgo, dev_conv2_works, conv2_worksSize, &beta, conv2_Tensor, dev_conv2_Output);
		hipdnnAddTensor(cudnn, &alpha, conv2_biasTensor, dev_conv2_Bias, &alpha, conv2_Tensor, dev_conv2_Output);
		hipdnnActivationForward(cudnn, conv2_Act_Desc, &alpha, conv2_Tensor, dev_conv2_Output, &beta, conv2_Tensor, dev_conv2Act_output);
		hipdnnPoolingForward(cudnn, pool2_Desc, &alpha, conv2_Tensor, dev_conv2Act_output, &beta, pool2_Tensor, dev_pool2_Output);
		hipdnnActivationForward(cudnn, conv2pool_Act_Desc, &alpha, pool2_Tensor, dev_pool2_Output, &beta, pool2_Tensor, dev_conv2pool_Act_output);
		//fc1 forward
		hipdnnConvolutionForward(cudnn, &alpha, pool2_Tensor, dev_conv2pool_Act_output, fc1_Filter_Desc, dev_fc1_Filter, fc1_Desc, fc1_fwAlgo, dev_fc1_works, fc1_worksSize, &beta, fc1_Tensor, dev_fc1_Output);
		hipdnnAddTensor(cudnn, &alpha, fc1_biasTensor, dev_fc1_Bias, &alpha, fc1_Tensor, dev_fc1_Output);
		hipdnnActivationForward(cudnn, fc1_Act_Desc, &alpha, fc1_Tensor, dev_fc1_Output, &beta, fc1_Tensor, dev_fc1_Actout);
		//fc2 forward
		hipdnnConvolutionForward(cudnn, &alpha, fc1_Tensor, dev_fc1_Actout, fc2_Filter_Desc, dev_fc2_Filter, fc2_Desc, fc2_fwAlgo, dev_fc2_works, fc2_worksSize, &beta, fc2_Tensor, dev_fc2_Output);
		hipdnnAddTensor(cudnn, &alpha, fc2_biasTensor, dev_fc2_Bias, &alpha, fc2_Tensor, dev_fc2_Output);
		hipdnnActivationForward(cudnn, fc2_Act_Desc, &alpha, fc2_Tensor, dev_fc2_Output, &beta, fc2_Tensor, dev_fc2_Actout);
		//fc3 forward
		hipdnnConvolutionForward(cudnn, &alpha, fc2_Tensor, dev_fc2_Actout, fc3_Filter_Desc, dev_fc3_Filter, fc3_Desc, fc3_fwAlgo, dev_fc3_works, fc3_worksSize, &beta, fc3_Tensor, dev_fc3_Output);
		hipdnnAddTensor(cudnn, &alpha, fc3_biasTensor, dev_fc3_Bias, &alpha, fc3_Tensor, dev_fc3_Output);
		hipdnnSoftmaxForward(cudnn, sftAlgo, sftMode, &alpha, fc3_Tensor, dev_fc3_Output, &beta, fc3_Tensor, dev_smaxOutput);
		//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
		hipMemcpy(yhat, dev_smaxOutput, sizeof(float) * batch_size_test * num_labels, hipMemcpyDeviceToHost);

		//one hot ��ġ ã��


		for (size_t i = 0; i < batch_size_test; i++) {
			float temp = yhat[i * 10];// ������ ù��° ���� �ӽ� ������ ����
			int indexJ = 0; // ���� ū���� ã�� ���� ��� �Ǵ� ��ġ �ε���

			for (size_t j = 0; j < num_labels - 1; j++) {
				if (temp > yhat[i * 10 + j + 1]) // �ӽ� ������ �־��� ���� ��
				{
					yhat[i * 10 + j + 1] = 0; // �ӽ� ������ ��� �ִ� ������ �۴ٸ� 0 �Է�
				}
				else                      // �ӽ� ������ ��� �ִ� ������ ũ�ٸ�
				{
					temp = yhat[i * 10 + j + 1]; // �ӽ� ������ �ش� ���� ����
					yhat[i * 10 + indexJ] = 0; // �ӽ� ������ ������ ��� �ִ� ���� �ε����� �̿��Ͽ� ���� �� ��ġ�� 0 �Է�
					indexJ = j + 1; // ���� ū ��(���� ���� ��) ��ġ�� �ε��� ������ ����
				}
			}

			predicted[i] = indexJ; // �ش� �̹����� ����Ʈ �ƽ� ���� ���� ū ���� ���� ��ġ �ε����� �迭�� ����
		}

		//accuracy ���

		for (size_t i = 0; i < batch_size_test; i++) {
			if (predicted[i] == target_test_batch[i])
				count++;
		}


	}


	std::cout << "================== �н� ����=================" << endl;

	cout << " �н� ��� �ð�             :: " << setw(10) << ((endTime_train - startTime_train) / ((CLOCKS_PER_SEC) * 60)) << " ��" << endl;
	cout << " �н��� ���� �̹��� ��    :: " << setw(10) << ImageNum << " ��" << endl;
	cout << " ��ġ ũ��(batch size)      :: " << setw(10) << batchSize << " ��" << endl;
	cout << " �н� ȸ��(epoch)           :: " << setw(10) << epoch << " ȸ" << endl;
	cout << " �н��� (learning_rate)     :: " << setw(10) << learning_rate << endl;
	cout << " ����� (momentum)     :: " << setw(10) << momentum << endl;


	cout << endl; cout << endl;

	std::cout << "=============== �׽�Ʈ ���� ===============" << endl;
	cout << " �׽�Ʈ�� ���� �̹��� ��   :: " << setw(10) << ImageNum_test << " ��" << endl;
	cout << " ��ġ ũ��(batch size)       :: " << setw(10) << batch_size_test << " ȸ" << endl;
	cout << endl; cout << endl;

	std::cout << "=============== �׽�Ʈ ��� ===============" << endl;
	cout << setw(2) << ImageNum_test << " ���� �̹����� �� " << count << " �� ����" << endl;
	cout << "��Ȯ��(Accuracy)     :: " << (count*1.0 / ImageNum_test) * 100 << " %" << endl;
	cout << "����(Error)          :: " << (1 - (count*1.0 / ImageNum_test)) * 100 << " %" << endl;
	cout << endl; cout << endl;

	std::cout << "��==========================================" << endl;

	hipFree(dev_bwf_conv1_Output);
	hipFree(dev_bw_conv1Bias);
	hipFree(dev_bw_conv1_ActDelta);
	hipFree(dev_pool1_Delta);
	hipFree(dev_bwd_fc3_Output);

	hipFree(dev_bwf_fc3_Output);
	hipFree(dev_bw_fc3Bias);


	hipFree(dev_smaxOutput);
	hipFree(dev_fc3_Bias);

	hipFree(dev_fc3_Output);
	hipFree(dev_fc3_Filter);
	hipFree(dev_pool1_Output);
	hipFree(dev_conv1Act_output);
	hipFree(dev_conv1_Bias);

	hipFree(dev_conv1_Output);
	hipFree(dev_conv1_Filter);
	hipFree(dev_input);



}