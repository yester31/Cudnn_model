#include "hip/hip_runtime.h"
#include <iomanip>
#include <iostream>
#include <cstdlib>
#include <math.h>
#include <hip/hip_runtime.h>
#include <hipDNN.h>
#include <stdio.h>
#include <string.h>
#include <opencv2/opencv.hpp>
#include <io.h>
#include <vector>


//***********************************************
//**���Ŀ� �� �������� GPU �޸� SizeȮ�� �ʿ�**
//***********************************************

//***********************
//**�ɼ� ���� ��� �߰�**
//***********************

//********************
//**alpha, beta ����**
//********************

using namespace std;
using namespace cv;

vector<pair<Mat, string>> TraverseFilesUsingDFS(const string& folder_path)
{
	_finddata_t file_info;
	string any_file_pattern = folder_path + "\\*";
	intptr_t handle = _findfirst(any_file_pattern.c_str(), &file_info);
	vector<pair<Mat, string>> ImgBox;


	//If folder_path exsist, using any_file_pattern will find at least two files "." and "..",
	//of which "." means current dir and ".." means parent dir
	if (handle == -1)
	{
		cerr << "folder path not exist: " << folder_path << endl;
		exit(-1);
	}

	//iteratively check each file or sub_directory in current folder
	do
	{
		string file_name = file_info.name; //from char array to string

										   //check whtether it is a sub direcotry or a file
		if (file_info.attrib & _A_SUBDIR)
		{
			if (file_name != "." && file_name != "..")
			{
				string sub_folder_path = folder_path + "\\" + file_name;
				TraverseFilesUsingDFS(sub_folder_path);
				cout << "a sub_folder path: " << sub_folder_path << endl;
			}
		}
		else  //cout << "file name: " << file_name << endl;
		{
			size_t npo1 = file_name.find('_') + 1;
			size_t npo2 = file_name.find('.');
			size_t npo3 = npo2 - npo1;
			string newname = file_name.substr(npo1, npo3);
			string sub_folder_path2 = folder_path + "\\" + file_name;
			Mat img = imread(sub_folder_path2);

			ImgBox.push_back({ { img },{ newname } });
		}
	} while (_findnext(handle, &file_info) == 0);

	//
	_findclose(handle);
	return ImgBox;
}


int main()
{

	const int numImgs = 100; // �̹��� �� ����
	string folder_path = "D:\\DataSet\\cifar\\test"; // �̹����� ����Ǿ� �ִ� ���� ���
	vector<pair<Mat, string>> ImgBox; // �̹��� ������, �̹��� �̸� 
	ImgBox = TraverseFilesUsingDFS(folder_path);
	vector<string> LabelBox; // �� ������ ���� ����
	vector<pair<int, string>> LabelTable; // �󺧸� ���� �ѹ� �ο�
	vector<pair<Mat, int>> ImgBox2; // �̹��� ������, �� �ѹ�
	vector<vector<int>> TargetY; // �� �ѹ� -> ������ ���� �����ͷ� ����

								 // �󺧿� ��ȣ �ο��� ���� LabelBox ���Ϳ� �� ���� �ϰ� ���� �� �ߺ� ����
	for (int i = 0; i < numImgs; i++)
	{
		LabelBox.push_back(ImgBox[i].second);
	}
	sort(LabelBox.begin(), LabelBox.end());
	LabelBox.erase(unique(LabelBox.begin(), LabelBox.end()), LabelBox.end());
	int nLabelBoxSize = LabelBox.size();

	// �� ��ȣ �ο�
	for (int i = 0; i < nLabelBoxSize; i++)
	{
		LabelTable.push_back({ { i },{ LabelBox[i] } });
	}


	//ImgBox2 ����
	for (int i = 0; i < numImgs; i++)
	{
		ImgBox2.push_back({ ImgBox[i].first, 0 });

		for (int j = 0; j < LabelTable.size(); j++)
		{
			if (ImgBox[i].second == LabelTable[j].second)
			{
				ImgBox2[i].second = LabelTable[j].first;
			}
		}
	}

	// TargetY ����, ���� ������ ���·� ǥ��
	TargetY.resize(numImgs);
	for (int i = 0; i < numImgs; i++)
	{
		TargetY[i].resize(nLabelBoxSize, 0);
	}
	for (int i = 0; i < numImgs; i++)
	{
		int idx = ImgBox2[i].second;
		TargetY[i][idx] = 1;
	}



	// 4�� ��� ���� �Ҵ� ����.
	int **** Input = new int***[numImgs];
	for (int i = 0; i < numImgs; i++)
	{
		Input[i] = new int**[3];
		for (int j = 0; j < 3; j++)
		{
			Input[i][j] = new int*[32];
			for (int k = 0; k < 32; k++)
			{
				Input[i][j][k] = new int[32];
			}
		}
	}

	// mat ���� - > 4�� ��� 
	for (int i = 0; i < numImgs; i++)
	{
		unsigned char* temp = ImgBox2[i].first.data;
		for (int c = 0; c < 3; c++)
		{
			for (int y = 0; y < 32; y++)
			{
				for (int x = 0; x < 32; x++)
				{
					Input[i][c][y][x] = temp[3 * 32 * y + 3 * x + c];
				}
			}
		}
	}



	//**********
	//**Handle**
	//**********
	hipdnnHandle_t cudnn;
	hipdnnCreate(&cudnn);

	//********
	//**����**
	//********

	//�Էº���
	const int ImageNum = numImgs;
	const int FeatureNum = 3;
	const int FeatureHeight = 32;
	const int FeatureWidth = 32;

	//********
	//**�Է�**
	//********




	//GPU�� �Է���� �޸� �Ҵ� �� �� ����
	float * dev_Input;
	hipMalloc((void**)&dev_Input, sizeof(Input));
	hipMemcpy(dev_Input, Input, sizeof(Input), hipMemcpyHostToDevice);

	//�Է���� ����ü ����, �Ҵ�, �ʱ�ȭ
	hipdnnTensorDescriptor_t in_desc; //�Է� ������ �� ������ ���� �ִ� ����ü�� ����Ű�� ���� ������
	hipdnnCreateTensorDescriptor(&in_desc); // 4D tensor ����ü ��ü ����
	hipdnnSetTensor4dDescriptor( // 4D tensor ����ü �ʱ�ȭ �Լ�
		/*tensorDesc,*/ in_desc,
		/*format,*/HIPDNN_TENSOR_NCHW,
		/*dataType,*/HIPDNN_DATA_FLOAT,
		/*Number of images*/ImageNum,
		/*C*/FeatureNum,
		/*H*/FeatureHeight,
		/*W*/FeatureWidth);

	//************************
	//************************
	//**Feedforward �������**
	//************************
	//************************

	//*******************
	//**Convolution����**
	//*******************

	//���� ������ ���� - ���� ���� �����ϵ���
	const int filt_n = 3;
	const int filt_c = 3;
	const int filt_h = 8;
	const int filt_w = 8;


	//���� ����
	float Filter[filt_n][filt_c][filt_h][filt_w];

	//���� ����
	int miner = -1;
	for (int och = 0; och < filt_n; och++)
	{
		for (int ch = 0; ch < filt_c; ch++)
		{
			for (int row = 0; row < filt_h; row++)
			{
				for (int col = 0; col < filt_w; col++)
				{


					Filter[och][ch][col][row] = (float)((col + row) % 3) *miner;
					miner *= -1;
				}
			}
		}
	}
	//GPU�� ������� ����
	float * dev_Filt;
	hipMalloc((void**)&dev_Filt, sizeof(float) * filt_n * filt_c * filt_h * filt_w);
	hipMemcpy(dev_Filt, Filter, sizeof(float) * filt_n * filt_c * filt_h * filt_w, hipMemcpyHostToDevice);

	//���ͱ���ü ����, ����, �ʱ�ȭ
	hipdnnFilterDescriptor_t filt_desc; // ���� ������ ���� ����ü�� ����Ű�� ���� ������
	hipdnnCreateFilterDescriptor(&filt_desc); // ���� ����ü ���� 
	hipdnnSetFilter4dDescriptor( // 4d filter ����ü ��ü �ʱ�ȭ
		/*filterDesc,*/filt_desc,
		/*dataType,*/HIPDNN_DATA_FLOAT,
		/*format,*/HIPDNN_TENSOR_NCHW,
		/*Number of output feature maps*/filt_n,
		/*Number of input feature maps.*/filt_c,
		/*Height of each filter.*/filt_h,
		/*Width of each filter.*/filt_w);



	//Convolution ���꿡���� ���� ���� - ���� ���� �����ϵ���
	const int pad_h = 2; //padding ����
	const int pad_w = 2; //padding ����
	const int str_h = 4; //stride ����
	const int str_w = 4; //stride ����
	const int dil_h = 1; //dilated ����
	const int dil_w = 1; //dilated ����

						 //Convolution ����ü ���� �� �Ҵ�
	hipdnnConvolutionDescriptor_t conv_desc; // Convolution ������ ���� ������ ���� ����ü ������ 
	hipdnnCreateConvolutionDescriptor(&conv_desc); // Convolution ����ü ��ü ����
	hipdnnSetConvolution2dDescriptor(//
		/*convDesc,*/conv_desc,
		/*zero-padding height*/pad_h,
		/*zero-padding width*/pad_w,
		/*Vertical filter stride*/str_h,
		/*Horizontal filter stride*/str_w,
		/*Filter height dilation*/dil_h,
		/*Filter width dilation*/dil_w,
		/*mode*/HIPDNN_CONVOLUTION,
		/*computeType*/HIPDNN_DATA_FLOAT);


	//Convolution ��� ������� ���� �� �Ҵ�
	hipdnnTensorDescriptor_t out_conv_desc;
	hipdnnCreateTensorDescriptor(&out_conv_desc);

	//Convolution ������ ������ ����
	int out_conv_n;
	int out_conv_c;
	int out_conv_h;
	int out_conv_w;

	hipdnnGetConvolution2dForwardOutputDim( // �־��� ����, tensor, convolution ����ü ������ ��������, 2D convolution ��꿡 ���� 4d tensor�� ��� ���� ������ ��ȯ, �� ��� output�� ���� �� �ε�...  
		/*convolution descriptor*/conv_desc,
		/*tensor descriptor*/in_desc,
		/*filter descriptor*/filt_desc,
		/*Output. Number of output images*/&out_conv_n,
		/*Output. Number of output feature maps per image.*/&out_conv_c,
		/*Output. Height of each output feature map.*/&out_conv_h,
		/*Output. Width of each output feature map.*/&out_conv_w);

	//outputDim = 1 + ( inputDim + 2*pad - (((filterDim-1)*dilation)+1) )/convolutionStride

	int outputDim = 1 + (FeatureHeight + 2 * pad_h - filt_h) / str_h;
	const int outputDimHW = 8;

	//Convolution��� ����
	float Output_Conv[ImageNum][FeatureNum][outputDimHW][outputDimHW];


	//GPU�� Convolution ��� ��� �Ҵ�
	float * dev_Output_Conv;
	hipMalloc((void**)&dev_Output_Conv, sizeof(float) * out_conv_c * out_conv_h * out_conv_n * out_conv_w);

	//Convolution ����ü �ʱ�ȭ
	hipdnnSetTensor4dDescriptor(out_conv_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT,
		ImageNum, FeatureNum, outputDim, outputDim);

	//�Է°� ����, ������� �е�, ��Ʈ���̵尡 ���� ���� �־������� ���� ���� �˰����� ���������� �˾Ƴ���
	hipdnnConvolutionFwdAlgo_t alg;
	alg = HIPDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_GEMM;

	//Conv ���� ������ũ�� �˾Ƴ��� �� �� ���� �޸� �Ҵ� �߰�
	size_t WS_size = 0;
	hipdnnGetConvolutionForwardWorkspaceSize(// This function returns the amount of GPU memory workspace
		cudnn, in_desc, filt_desc, conv_desc, out_conv_desc, alg, &WS_size);

	size_t * dev_WS;
	hipMalloc((void**)&dev_WS, WS_size);

	//����
	float alpha = 1.0;
	float beta = 0.0;

	hipdnnConvolutionForward(
		cudnn,
		&alpha,
		in_desc,
		dev_Input,
		filt_desc,
		dev_Filt,
		conv_desc,
		alg,
		dev_WS,
		WS_size,
		&beta,
		out_conv_desc,
		dev_Output_Conv);

	//Convolution��� GPU�� ����
	hipMemcpy(Output_Conv, dev_Output_Conv,
		sizeof(float) * out_conv_n * out_conv_c * out_conv_h * out_conv_w, hipMemcpyDeviceToHost);



	//********
	//**Bias**
	//********
	beta = 1.0f;

	//Bias ��� ������� ����
	float Output_Bias[ImageNum][FeatureNum][outputDimHW][outputDimHW];

	//bias �� ����
	float biasValue[filt_n] = { -10.0f };

	//GPU�� bias�� ����
	float * dev_Bias;
	hipMalloc((void**)&dev_Bias, sizeof(float));
	hipMemcpy(dev_Bias, biasValue, sizeof(float), hipMemcpyHostToDevice);

	//bias��� ������� ����, �Ҵ�
	hipdnnTensorDescriptor_t bias_desc;
	hipdnnCreateTensorDescriptor(&bias_desc);
	hipdnnSetTensor4dDescriptor(
		bias_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, 1, filt_n, 1, 1);

	//bias ���� ���� 
	hipdnnAddTensor(cudnn, &alpha, bias_desc, dev_Bias,
		&beta, /*input -> output*/out_conv_desc, /*input -> output*/dev_Output_Conv);

	//Bias�� ���
	hipMemcpy(Output_Bias, dev_Output_Conv,
		sizeof(float) * ImageNum * FeatureNum * outputDimHW * outputDimHW, hipMemcpyDeviceToHost);


	//***********************
	//**Actiovation Funtion**
	//***********************

	beta = 0.0;

	//Activation Function ����ü ���� �� �Ҵ� 
	hipdnnActivationDescriptor_t act_desc;
	hipdnnCreateActivationDescriptor(&act_desc);

	//Activation Function ���� ���� - ���� ���������ϵ���
	hipdnnActivationMode_t Activation_Function;
	//Activation_Function = HIPDNN_ACTIVATION_RELU;
	//Activation_Function = HIPDNN_ACTIVATION_TANH; 
	Activation_Function = HIPDNN_ACTIVATION_SIGMOID;

	hipdnnSetActivationDescriptor(act_desc, Activation_Function, HIPDNN_PROPAGATE_NAN, 0);

	//Activation Function �޸� GPU�� ����
	float * dev_Output_Act;
	hipMalloc((void**)&dev_Output_Act, sizeof(float) * outputDimHW * outputDimHW * 3);


	//Activatin Function �������
	hipdnnActivationForward(
		cudnn, act_desc, &alpha, out_conv_desc, dev_Output_Conv,
		&beta, out_conv_desc, dev_Output_Act);

	//Activation Function ����� ���� ���
	float Output_Activation[ImageNum][FeatureNum][outputDimHW][outputDimHW];
	hipMemcpy(Output_Activation, dev_Output_Act, sizeof(float) * ImageNum * FeatureNum * outputDimHW * outputDimHW, hipMemcpyDeviceToHost);

	//Actavation Function ���

	//***************
	//**Pooling����**
	//***************

	//Pooling ���꿡�� ���� ���� - ���� ���� �����ϵ���
	beta = 0.0;

	const int pool_wind_h = 2;
	const int pool_wind_w = 2;
	const int pool_pad_h = 0;
	const int pool_pad_w = 0;
	const int pool_strd_w = 2;
	const int pool_strd_h = 2;

	//Pooling ����ü ���� �� �Ҵ� - ���� Pooling ��� ���� �����ϵ���
	hipdnnPoolingDescriptor_t pool_desc;
	hipdnnCreatePoolingDescriptor(&pool_desc);
	hipdnnSetPooling2dDescriptor(pool_desc, HIPDNN_POOLING_MAX, HIPDNN_PROPAGATE_NAN,
		pool_wind_h, pool_wind_w, pool_pad_h, pool_pad_w, pool_strd_h, pool_strd_w);

	//Pooling ������� ��� ���� �� �Ҵ�
	hipdnnTensorDescriptor_t out_pool_desc;
	hipdnnCreateTensorDescriptor(&out_pool_desc);

	//Pooling ������ ������
	int out_pool_n;
	int out_pool_c;
	int out_pool_h;
	int out_pool_w;

	//Pooling ������ ������ ����
	hipdnnGetPooling2dForwardOutputDim(pool_desc, out_conv_desc,
		&out_pool_n, &out_pool_c, &out_pool_h, &out_pool_w);

	//GPU�� Pooling ������ �޸��Ҵ�
	float * dev_Output_Pool;
	hipMalloc((void**)&dev_Output_Pool,
		sizeof(float) * out_pool_n * out_pool_c * out_pool_h * out_pool_w);

	//Pooling ������� ����ü �ʱ�ȭ
	hipdnnSetTensor4dDescriptor(out_pool_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT,
		out_pool_n, out_pool_c, out_pool_h, out_pool_w);

	//Pooling���� ����
	hipdnnPoolingForward(cudnn, pool_desc, &alpha, out_conv_desc, dev_Output_Act,
		&beta, out_pool_desc, dev_Output_Pool);

	//Pooling���
	float Output_Pool[ImageNum][FeatureNum][(outputDimHW + 2 * pool_pad_h) / pool_strd_h][(outputDimHW + 2 * pool_pad_w) / pool_strd_w];
	hipMemcpy(Output_Pool, dev_Output_Pool,
		sizeof(float) * out_pool_n * out_pool_c * out_pool_h * out_pool_w, hipMemcpyDeviceToHost);

	//Pooling��� 


	//*******************
	//**Fully Connected**
	//*******************

	//Weights ����
	float Weights[10][3][4][4];

	//Weights ����
	for (int och = 0; och < 10; och++)
	{
		for (int ch = 0; ch < 3; ch++)
		{
			for (int row = 0; row < 4; row++)
			{
				for (int col = 0; col < 4; col++)
				{
					Weights[och][ch][row][col] = (float)(row + col + och + ch)*0.11;
				}
			}
		}
	}


	//GPU�� Weights��� ����
	float * dev_weights;
	hipMalloc((void**)&dev_weights, sizeof(float) * 10 * 3 * 4 * 4);
	hipMemcpy(dev_weights, Weights, sizeof(float) * 10 * 3 * 4 * 4, hipMemcpyHostToDevice);

	//Weights�� ���� Filter ����ü ���� �� �Ҵ�
	hipdnnFilterDescriptor_t weights_desc;
	hipdnnCreateFilterDescriptor(&weights_desc);
	hipdnnSetFilter4dDescriptor(weights_desc, HIPDNN_DATA_FLOAT, HIPDNN_TENSOR_NCHW, 10, 3, 4, 4);

	//Fully Connected�� ���� Convolution ����ü ���� �� �Ҵ�
	hipdnnConvolutionDescriptor_t fc_desc;
	hipdnnCreateConvolutionDescriptor(&fc_desc);
	hipdnnSetConvolution2dDescriptor(fc_desc, 0, 0, 1, 1, 1, 1, HIPDNN_CONVOLUTION, HIPDNN_DATA_FLOAT);

	//Fully Connected ���� ��� ������� ����ü ����
	hipdnnTensorDescriptor_t out_fc_desc;
	hipdnnCreateTensorDescriptor(&out_fc_desc);

	//Fully Connected ������ ������ ����
	int out_fc_n;
	int out_fc_c;
	int out_fc_h;
	int out_fc_w;

	hipdnnGetConvolution2dForwardOutputDim(
		fc_desc, out_fc_desc, weights_desc, &out_fc_n, &out_fc_c, &out_fc_h, &out_fc_w);

	//FC ������ ����
	float Output_FC[ImageNum][10][1][1];

	//GPU�� FC ������ �Ҵ�
	float *dev_Output_FC;
	hipMalloc((void**)&dev_Output_FC, sizeof(float) * ImageNum * 10 * 1 * 1);

	//FC ����ü �ʱ�ȭ
	hipdnnSetTensor4dDescriptor(out_fc_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, ImageNum, 10, 1, 1);


	//FC ����ũ�� �Ҵ� �� ����
	size_t WS_size2 = 0;
	hipdnnGetConvolutionForwardWorkspaceSize(
		cudnn, out_pool_desc, weights_desc, fc_desc, out_fc_desc, alg, &WS_size2);

	size_t * dev_WS2;
	hipMalloc((void**)&dev_WS2, WS_size2);

	//Fully Connected ���� 
	hipdnnConvolutionForward(
		cudnn, &alpha, out_pool_desc, dev_Output_Pool, weights_desc, dev_weights, fc_desc,
		alg, dev_WS2, WS_size2, &beta, out_fc_desc, dev_Output_FC);

	//FC ����� CPU�� ����
	hipMemcpy(Output_FC, dev_Output_FC, sizeof(float) * ImageNum * 10 * 1 * 1, hipMemcpyDeviceToHost);



	//*************************
	//**Fully Conncected Bias**
	//*************************
	beta = 1.0f;

	//FC bias ��� ����
	float Output_FC_Bias[ImageNum][10][1][1];

	//FC bias��
	float biasValueFC[1] = { -5.0f };

	//GPU�� FC bias�� ����
	float * dev_Bias_FC;
	hipMalloc((void**)&dev_Bias_FC, sizeof(float));
	hipMemcpy(dev_Bias_FC, biasValueFC, sizeof(float), hipMemcpyHostToDevice);


	//FC Softmax ����ü - �� �� ��������?
	hipdnnTensorDescriptor_t out_Bias_FC_desc;
	hipdnnCreateTensorDescriptor(&out_Bias_FC_desc);
	hipdnnSetTensor4dDescriptor(out_Bias_FC_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, ImageNum, 10, 1, 1);


	//bias ���� ����
	hipdnnAddTensor(cudnn, &alpha, out_Bias_FC_desc, dev_Bias_FC, &beta, out_fc_desc, dev_Output_FC);
	hipMemcpy(Output_FC_Bias, dev_Output_FC, sizeof(float) * ImageNum * 10, hipMemcpyDeviceToHost);


	//***********
	//**Softmax**
	//***********
	beta = 0.0;

	float OutSoft[ImageNum][10][1][1];
	float * dev_Output_Softmax;
	hipMalloc((void**)&dev_Output_Softmax, sizeof(float) * ImageNum * 10);


	hipdnnSoftmaxForward(cudnn, HIPDNN_SOFTMAX_FAST, HIPDNN_SOFTMAX_MODE_INSTANCE/*�� �κ� �ſ� �߿� - INSTANCE�� �ؾ� �ٷ� �̺а��*/,
		&alpha, out_fc_desc, dev_Output_FC, &beta, out_fc_desc, dev_Output_Softmax);

	hipMemcpy(OutSoft, dev_Output_Softmax, sizeof(float) * ImageNum * 10, hipMemcpyDeviceToHost);

	//*********
	//**Error**
	//*********
	vector<float> error;

	error.resize(ImageNum);
	float dy[ImageNum][10][1][1];



	// cost function (ũ�ν� ��Ʈ����)���� ���� ��� 
	float sum = 0;
	for (int n = 0; n < ImageNum; n++)
	{
		for (int c = 0; c < 10; c++) {
			
			sum += (-log(OutSoft[n][c][0][0]) * TargetY[n][c]);
			dy[n][c][0][0] = (TargetY[n][c] - OutSoft[n][c][0][0]);
		}
		error[n] = sum;
		sum = 0;
	}

	/*

	//Cross Entropy, ���� ���
	std::cout << std::endl << std::endl << "Cross Entropy ��" << std::endl << std::endl;

	for (int n = 0; n < ImageNum; n++)
	{
		std::cout << n << " :: " << error[n];
		std::cout << std::endl;
	}

	std::cout << std::endl;
	*/


	//****************************
	//****************************
	//**Backpropagation �������**
	//****************************
	//****************************

	//***************************
	//**Softmax Backpropagation** - p - y
	//***************************



	//������� ����
	float SoftBack[ImageNum][10][1][1];

	//GPU �޸� �Ҵ�
	float * dif_Soft_Back;
	hipMalloc((void**)&dif_Soft_Back, sizeof(float) * ImageNum * 10);

	//����ü ���� �� �ʱ�ȭ
	hipdnnTensorDescriptor_t dif_soft_desc;
	hipdnnCreateTensorDescriptor(&dif_soft_desc);
	hipdnnSetTensor4dDescriptor(dif_soft_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, ImageNum, 10, 1, 1);


	//delta

	float * dev_Output_Soft_Back;
	hipMalloc((void**)&dev_Output_Soft_Back, sizeof(float) * ImageNum * 10);



	float * dev_dif_Softmax;
	hipMalloc((void**)&dev_dif_Softmax, sizeof(float) * ImageNum * 10);
	hipMemcpy(dev_dif_Softmax, dy, sizeof(float) * ImageNum * 10, hipMemcpyHostToDevice);


	hipdnnTensorDescriptor_t dif_soft_back;
	hipdnnCreateTensorDescriptor(&dif_soft_back);
	hipdnnSetTensor4dDescriptor(dif_soft_back, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, ImageNum, 10, 1, 1);



	hipdnnSoftmaxBackward(cudnn, HIPDNN_SOFTMAX_ACCURATE, /*****�̺κ� �ſ� �߿�*****/HIPDNN_SOFTMAX_MODE_INSTANCE, &alpha,
		dif_soft_back, dev_Output_Softmax, dif_soft_back, dev_dif_Softmax, &beta, dif_soft_back, dev_Output_Soft_Back);

	hipMemcpy(SoftBack, dev_Output_Soft_Back, sizeof(float) * ImageNum * 10, hipMemcpyDeviceToHost);

	for (int n = 0; n < ImageNum; n++)
	{
	for (int i = 0; i < 10; i++) {

		std::cout << SoftBack[n][i][0][0] << "  ::   " << dy[n][i][0][0] <<std::endl;
	}
	std::cout << std::endl;
	}

	//*********************************
	//**Fully Connected Bias Backward**
	//*********************************

	

	//�������
	float FCbiasBack[ImageNum][10][1][1];

	//GPU �޸�
	float * dev_FC_bias_Back;
	hipMalloc((void**)&dev_FC_bias_Back, sizeof(float));

	hipdnnConvolutionBackwardBias(cudnn, &alpha, out_fc_desc, dev_Output_Softmax, &beta, bias_desc, dev_FC_bias_Back);

	hipMemcpy(FCbiasBack, dev_FC_bias_Back, sizeof(float), hipMemcpyDeviceToHost);

	for (int i = 0; i < 10; i++) {
	
		std::cout << FCbiasBack[0][i][1][1];
	}



	//**********************************
	//**Fully Connected Backpropagtion**
	//**********************************

	//������� ����
	float FCBack[3][10][32][32];

	//GPU�� �޸� �Ҵ�
	float * dev_Filter_Gradient;
	hipMalloc((void**)&dev_Filter_Gradient, sizeof(float) * 3 * 10 * 32 * 32);

	// Workspace
	size_t WS_size3 = 0;
	hipdnnGetConvolutionBackwardDataWorkspaceSize(cudnn, weights_desc, dif_soft_back, fc_desc, out_pool_desc, HIPDNN_CONVOLUTION_BWD_DATA_ALGO_0, &WS_size3);

	//GPU�� workspace �޸� �Ҵ�
	size_t * dev_WS3;
	hipMalloc((void**)&dev_WS3, WS_size3);

	//Fully Connected Backpropagation delta
	hipdnnConvolutionBackwardFilter(cudnn, &alpha,
		out_pool_desc, dev_Output_Pool, dif_soft_back, dev_Output_Soft_Back, fc_desc, HIPDNN_CONVOLUTION_BWD_FILTER_ALGO_0,
		dev_WS3, WS_size3, &beta, weights_desc, dev_Filter_Gradient);

	//CPU�� ��� ����
	hipMemcpy(FCBack, dev_Filter_Gradient, sizeof(float) * 3 * 10 * 32 * 32, hipMemcpyDeviceToHost);


	/*
	std::cout << std::endl << std::endl << "FCBack" << std::endl << std::endl;

	for (int i = 0; i < 32; i++)
	{
		for (int j = 0; j < 32; j++)
		{
			std::cout << setw(3) << FCBack[0][0][i][j] << " :: ";
		}
		std::cout << std::endl;
	}
	*/



}